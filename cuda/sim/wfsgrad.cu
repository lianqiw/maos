#include "hip/hip_runtime.h"
/*
  Copyright 2009-2025 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#define TIMING 0
#include <hiprand/hiprand_kernel.h>
#include "../math/cumath.h"
#include "accphi.h"
#include "wfs.h"
#include "cudata.h"
#if !USE_CPP
extern "C"
{
#endif
#include "../../maos/sim.h"
#include "../../maos/utils.h"
#if !USE_CPP
}
#endif

extern const char* dirskysim;
/*
  Notice that both blocks and threads are partitioning isa
*/
static __global__ void
add_geom_noise_do(Real* restrict g, const Real* restrict nea,
	int nsa, hiprandState* restrict rstat){
	const int id=threadIdx.x+blockIdx.x*blockDim.x;
	hiprandState lstat=rstat[id];
	const int nstep=blockDim.x*gridDim.x;
	for(int i=id; i<nsa; i+=nstep){
		Real n1=hiprand_normal(&lstat);
		Real n2=hiprand_normal(&lstat);
		g[i]+=n1*nea[i];
		g[i+nsa]+=n2*nea[i+nsa]+n1*nea[i+nsa*2];/*cross term. */
	}
	rstat[id]=lstat;
}

/**
   Compute ztilt over a square subaperture.
*/
static __global__ void
cuztilt_do(Real* restrict g, Real* restrict opd,
	const int nsa, const Real dx, const int nx, Real* imcc,
	const Real(*orig)[2], const Real* restrict amp, Real alpha){
	extern __shared__ Real a0[];
	int idx=threadIdx.x+threadIdx.y*blockDim.x;
	Real* a[3];
	for(int i=0; i<3; i++){
		a[i]=a0+blockDim.x*blockDim.y*i;
		a[i][idx]=0;
	}
	const int isa=blockIdx.x;
	const int skip=isa*nx*nx;
	const Real ox=orig[isa][0];
	const Real oy=orig[isa][1];
	for(int iy=threadIdx.y; iy<nx; iy+=blockDim.y){
		const int skip2=skip+iy*nx;
		const Real y=iy*dx+oy;
		for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
			const int ind=skip2+ix;
			const Real tmp=amp[ind]*opd[ind];
			a[0][idx]+=tmp;
			a[1][idx]+=tmp*(dx*ix+ox);
			a[2][idx]+=tmp*y;
		}
	}
	for(int step=(blockDim.x*blockDim.y)>>1;step>0;step>>=1){
		__syncthreads();
		if(idx<step){
			for(int i=0; i<3; i++){
				a[i][idx]+=a[i][idx+step];
			}
		}
	}
	__syncthreads();
	if(threadIdx.x<2&&threadIdx.y==0){
		Real(*restrict A)[3]=(Real(*)[3])(imcc+isa*9);
		atomicAdd(&g[isa+threadIdx.x*nsa],
			alpha*(+a[0][0]*A[0][threadIdx.x+1]
				   +a[1][0]*A[1][threadIdx.x+1]
				   +a[2][0]*A[2][threadIdx.x+1]));

	}
}
void cuztilt(Real* restrict g, Real* restrict opd,
	const int nsa, const Real dx, const int nx, Real* imcc,
	const Real(*orig)[2], const Real* restrict amp, Real alpha, hipStream_t stream){
	const int tx=32;
	cuztilt_do<<<nsa, dim3(tx, tx), tx* tx*3*sizeof(Real), stream>>>
		(g, opd, nsa, dx, nx, imcc, orig, amp, alpha);
}
/**
   Apply matched filter. \todo this implementation relies on shared variable. It
   is probably causing competition.  */
__global__ static void
mtche_do(Real* restrict grad, Real(*mtches)[2], const Real* restrict ints,
	int sigmatch, const Real* restrict i0sum, Real scale, int pixpsa, int nsa){
	extern __shared__ Real g0[];
	Real* g[3];
	for(int i=0; i<3; i++){
		g[i]=g0+blockDim.x*i;
		g[i][threadIdx.x]=0;
	}
	int isa=blockIdx.x;
	ints+=isa*pixpsa;
	const Real(* const restrict mtche)[2]=mtches+pixpsa*isa;

	for(int ipix=threadIdx.x; ipix<pixpsa; ipix+=blockDim.x){
		g[0][threadIdx.x]+=mtche[ipix][0]*ints[ipix];
		g[1][threadIdx.x]+=mtche[ipix][1]*ints[ipix];
		g[2][threadIdx.x]+=ints[ipix];
	}
	for(int step=(blockDim.x)>>1;step>0;step>>=1){
		__syncthreads();
		if(threadIdx.x<step){
			for(int i=0; i<3; i++){
				g[i][threadIdx.x]+=g[i][threadIdx.x+step];
			}
		}
	}
	__syncthreads();
	if(threadIdx.x<2){
		if(sigmatch==1){/*normalize gradients according to siglev.*/
			Real scale_isa=i0sum[isa]/g[2][0];
			if(!(scale_isa<10 && scale_isa > 0)) scale_isa=1;
			g[threadIdx.x][0]*=scale_isa;
		} else if(sigmatch==2){/*normalize gradients according to subaperture averaged siglev.*/
			g[threadIdx.x][0]*=scale;
		}//else: do not normalize
		grad[isa+nsa*threadIdx.x]=g[threadIdx.x][0];
	}
}

static void
mtche(Real* restrict grad, Real(*mtches)[2],
	const Real* restrict ints, int sigmatch, const Real* restrict i0sum, Real scale,
	int pixpsa, int nsa, int msa, hipStream_t stream){
	for(int isa=0; isa<nsa; isa+=msa){
		int ksa=MIN(msa, nsa-isa);
		mtche_do<<<ksa, 16, 16*3*sizeof(Real), stream>>>
			(grad+isa, mtches+pixpsa*isa, ints+pixpsa*isa, sigmatch, i0sum?i0sum+isa:0, scale, pixpsa, nsa);
	}
}
/**
   Apply windowed CoG. /todo: replace atomicAdd by reduction.
*/
__global__ static void
wcog_do(Real* grad, const Real* restrict ints, Real scale1, Real* scale2, 
	int nx, int ny, Real pixthetax, Real pixthetay, int nsa, Real* cogmask, Real bkgrnd, Real* srot){
	__shared__ Real sum[3];
	if(threadIdx.x<3&&threadIdx.y==0) sum[threadIdx.x]=0.f;
	__syncthreads();//is this necessary?
	int isa=blockIdx.x;
	ints+=isa*nx*ny;
	//loop can be replaced if block dimension matches nx, ny exactly.
	for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
		for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
			Real im=(ints[ix+iy*nx]-bkgrnd)*cogmask[ix+iy*nx];
			atomicAdd(&sum[0], im);
			atomicAdd(&sum[1], im*ix);
			atomicAdd(&sum[2], im*iy);
		}
	}
	__syncthreads();
	if(threadIdx.x==0&&threadIdx.y==0){
		if(scale2){
			Real sum2=scale1*scale2[isa];
			sum[0]=sum2; //replace intensity by constant.
		}
		if(sum[0]>0){
			Real gx=(sum[1]/sum[0]-(nx-1)*0.5)*pixthetax;
			Real gy=(sum[2]/sum[0]-(ny-1)*0.5)*pixthetay;
			if(srot){
				Real s, c;
				Z(sincos)(srot[isa], &s, &c);
				Real tmp=gx*c-gy*s;
				gy=gx*s+gy*c;
				gx=tmp;
			}
			grad[isa]=gx;
			grad[isa+nsa]=gy;
		} else{
			grad[isa]=0;
			grad[isa+nsa]=0;
		}
	}
}
/**
   Apply thresholded CoG. /todo: replace atomicAdd by reduction.
*/
__global__ static void
tcog_do(Real* grad, const Real* restrict ints, Real scale1, Real* scale2, 
	int nx, int ny, Real pixthetax, Real pixthetay, int nsa, Real thres, Real bkgrnd, Real* srot){
	__shared__ Real sum[3];
	if(threadIdx.x<3&&threadIdx.y==0) sum[threadIdx.x]=0.f;
	__syncthreads();//is this necessary?
	int isa=blockIdx.x;
	ints+=isa*nx*ny;
	//loop can be replaced if block dimension matches nx, ny exactly.
	for(int iy=threadIdx.y; iy<ny; iy+=blockDim.y){
		for(int ix=threadIdx.x; ix<nx; ix+=blockDim.x){
			Real im=ints[ix+iy*nx]-bkgrnd;
			if(im>thres){
				atomicAdd(&sum[0], im);
				atomicAdd(&sum[1], im*ix);
				atomicAdd(&sum[2], im*iy);
			}
		}
	}
	__syncthreads();
	if(threadIdx.x==0&&threadIdx.y==0){
		if(scale2){
			Real sum2=scale1*scale2[isa];
			sum[0]=sum2; //replace intensity by constant.
		}
		if(sum[0]>thres){
			Real gx=(sum[1]/sum[0]-(nx-1)*0.5)*pixthetax;
			Real gy=(sum[2]/sum[0]-(ny-1)*0.5)*pixthetay;
			if(srot){
				Real s, c;
				Z(sincos)(srot[isa], &s, &c);
				Real tmp=gx*c-gy*s;
				gy=gx*s+gy*c;
				gx=tmp;
			}
			grad[isa]=gx;
			grad[isa+nsa]=gy;
		} else{
			grad[isa]=0;
			grad[isa+nsa]=0;
		}
	}
}
/**
   Poisson random generator.
*/
__device__ static Real curandp(hiprandState* rstat, Real xm){
	Real g, t, xmu;
	int x=0, xu;
	if(xm>200){
		x=(int)round(xm+hiprand_normal(rstat)*sqrt(xm));
	} else{
		while(xm>0){
			xmu=xm>12.f?12.f:xm;
			xm-=xmu;
			g=__expf(-xmu);
			xu=-1;
			t=1.f;
			while(t>g){
				xu++;
				t*=hiprand_uniform(rstat);
			}
			x+=xu;
		}
	}
	return x;
}
/**
   Add noise to pix images.
*/
__global__ static void addnoise_do(Real* restrict ints0, int nsa, int pixpsa, Real bkgrnd, Real bkgrndc,
	const Real* restrict bkgrnd2s,
	const Real* restrict bkgrnd2cs,
	const Real* restrict qe, Real rne, hiprandState* rstat){
	const int id=threadIdx.x+blockIdx.x*blockDim.x;
	const int nstep=blockDim.x*gridDim.x;
	hiprandState lstat=rstat[id];
	for(int isa=id; isa<nsa; isa+=nstep){
		Real* restrict ints=ints0+isa*pixpsa;
		const Real* restrict bkgrnd2=bkgrnd2s?(bkgrnd2s+isa*pixpsa):NULL;
		const Real* restrict bkgrnd2c=bkgrnd2cs?(bkgrnd2cs+isa*pixpsa):NULL;
		for(int ipix=0; ipix<pixpsa; ipix++){
			Real tot=(ints[ipix]+bkgrnd+(bkgrnd2?bkgrnd2[ipix]:0));
			Real corr=bkgrnd2c?(bkgrnd2c[ipix]+bkgrndc):bkgrndc;
			if(qe){
				ints[ipix]=(curandp(&lstat, tot*qe[ipix])+rne*hiprand_normal(&lstat))/qe[ipix]-corr;
			} else{
				ints[ipix]=curandp(&lstat, tot)+rne*hiprand_normal(&lstat)-corr;
			}
		}
	}
	rstat[id]=lstat;
}
static void
cu_fieldstop(curmat& opd, curmat& amp, int* embed, int nembed,
	curmat& fieldstop, Real wvl, hipfftHandle fftplan, hipStream_t stream){
	cucmat wvf(nembed, nembed);
	embed_wvf_do<<<DIM(opd.Nx(), 256), 0, stream>>>(wvf, opd, amp, embed, opd.Nx(), wvl);
	CUFFT(fftplan, wvf, HIPFFT_FORWARD);
	cwm_do<<<DIM(wvf.N(), 256), 0, stream>>>(wvf(), fieldstop(), wvf.N());
	CUFFT(fftplan, wvf, HIPFFT_BACKWARD);
	unwrap_phase_do<<<DIM2(wvf.Nx(), wvf.Ny(), 16), 0, stream>>>(wvf, opd, embed, opd.Nx(), wvl);
}

__global__ static void
dither_acc_do(Real* restrict* imb, Real* restrict* imx, Real* restrict* imy,
	Real* restrict const* pints, Real cd, Real sd, int pixpsa, int nsa){
	for(int isa=blockIdx.x; isa<nsa; isa+=gridDim.x){
		const Real* ints=pints[isa];
		Real* restrict acc_ints=imb[isa];
		Real* restrict acc_intsx=imx[isa];
		Real* restrict acc_intsy=imy[isa];
		for(int ipix=threadIdx.x; ipix<pixpsa; ipix+=blockDim.x){
			acc_ints[ipix]+=ints[ipix];
			acc_intsx[ipix]+=ints[ipix]*cd;
			acc_intsy[ipix]+=ints[ipix]*sd;
		}
	}
}
Dither_t::Dither_t(int nsa, int pixpsax, int pixpsay, int xy):imc(0){
	imb=curcell(nsa, 1, pixpsax, pixpsay);
	if(xy){
		imx=curcell(nsa, 1, pixpsax, pixpsay);
		imy=curcell(nsa, 1, pixpsax, pixpsay);
	}
}

/**Accumulate for matched filter updating*/
void Dither_t::acc(dither_t* dither, curcell& ints, Real cs, Real ss, int npll, hipStream_t stream){
	const int nsa=ints.N();
	const int pixpsa=ints[0].N();
	dither_acc_do<<<nsa, pixpsa, 0, stream>>>
		(imb.pm, imx.pm, imy.pm, ints.pm, cs, ss, pixpsa, nsa);
	imc++;
	if(imc%npll==0){
		//dbg("Dither::acc: output imb, imx, imy\n");
		cp2cpu(&dither->imb, imb, stream);
		cp2cpu(&dither->imx, imx, stream);
		cp2cpu(&dither->imy, imy, stream);
		cuzero(imb);
		cuzero(imx);
		cuzero(imy);
	}
}

/**Accumulate for matched filter updating*/
void Dither_t::acc_i0(dither_t *dither, curcell &ints, int npll, hipStream_t stream){
	curcelladd(imb, 1, ints, 1, stream);
	imc++;
	if(imc%npll==0){
		//dbg("Dither::acc: output imb, imx, imy\n");
		cp2cpu(&dither->imb, imb, stream);
		cuzero(imb);
	}
}
/*
//The following does not work as memcpy is not allowed in callback.
struct wfsgrad_callback_t{
	sim_t *simu;
	int iwfs;
	int op;//operation
};
static void wfsgrad_callback(hipStream_t stream, hipError_t status, void *data_){
	struct wfsgrad_callback_t* data=(struct wfsgrad_callback_t*)data_;
	sim_t *simu=data->simu;
	Array<cuwfs_t>& cuwfs=cuglobal->wfs;
	const parms_t *parms=simu->parms;
	const int iwfs=data->iwfs;
	const int ipowfs=parms->wfs[iwfs].powfs;
	const int isim=simu->wfsisim;
	const int do_phy=(parms->powfs[ipowfs].usephy&&isim>=parms->powfs[ipowfs].phystep);
	const int dtrat=parms->powfs[ipowfs].dtrat;
	const int dtrat_output=((isim+1)%dtrat==0);
	const int phytype=parms->powfs[ipowfs].phytype_sim;
	info("wfsgrad_callback runs with op=%d, iwfs=%d\n", data->op, data->iwfs);
	CUDA_CHECK_ERROR;
	switch(data->op){
		case 1:{
			cp2cpu(&simu->ints->p[iwfs], cuwfs[iwfs].ints);
			shwfs_grad(&simu->gradcl->p[iwfs], P(simu->ints->p[iwfs]),
				parms, simu->powfs, iwfs, phytype);
		}break;
		case 2:{
			if(dtrat_output){
				if(do_phy){
					if(phytype<3||parms->powfs[ipowfs].type==WFS_PY){
						cp2cpu(&simu->gradcl->p[iwfs], cuwfs[iwfs].gradcalc);
					}
					if(parms->save.gradgeom->p[iwfs]){//also do geom grad during phy grad sims
						zfarr_push(simu->save->gradgeom[iwfs], simu->wfsisim, cuwfs[iwfs].gradacc);
					}
					if(parms->plot.run&&draw_current("Ints", NULL)){// && parms->powfs[ipowfs].lo){
						cp2cpu(&simu->ints->p[iwfs], cuwfs[iwfs].ints);
					}
				} else{
					cp2cpu(&simu->gradcl->p[iwfs], cuwfs[iwfs].gradacc);
				}
			}
		}break;
		default:
			error("Invalid op=%d\n", data->op);
	}
	free(data);
}*/
/**
   Calculate SHWFS gradients
*/
static void shwfs_grad(curmat& gradcalc, const curcell& ints, Array<cuwfs_t>& cuwfs, Array<cupowfs_t>& cupowfs,
	const parms_t* parms, const powfs_t* powfs, sim_t* simu, int iwfs, int ipowfs, stream_t& stream){
	const int nsa=powfs[ipowfs].saloc->nloc;
	const int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
	CUDA_CHECK_ERROR;
	//gradcalc.Zero(stream);//(no need, mtche, tcog_do does not accumulate.)
	const int totpix=powfs[ipowfs].pixpsax*powfs[ipowfs].pixpsay;
	//static int last_phytype=-1;
	const Real cogthres=parms->powfs[ipowfs].cogthres;
	const Real cogoff=parms->powfs[ipowfs].cogoff;
	switch(parms->powfs[ipowfs].phytype_sim){
	case 0:
		break; //no-op
	case PTYPE_MF://Matched filter
	{
		Real sigratio=parms->powfs[ipowfs].sigmatch==2?(cuwfs[iwfs].i0sumsum/cursum(ints.M(), stream)):0;
		mtche(gradcalc, (Real(*)[2])(cuwfs[iwfs].mtche()), ints.M(),
			parms->powfs[ipowfs].sigmatch, cuwfs[iwfs].i0sum(), sigratio,
			totpix, nsa, cuwfs[iwfs].msa, stream);
	}
	break;
	case PTYPE_COG:
	{//CoG
		Real pixthetax=(Real)parms->powfs[ipowfs].radpixtheta;
		Real pixthetay=(Real)parms->powfs[ipowfs].pixtheta;
		int pixpsax=powfs[ipowfs].pixpsax;
		int pixpsay=powfs[ipowfs].pixpsay;
		Real scale1=0;
		Real* scale2=0;
		switch(parms->powfs[ipowfs].sigmatch){
		case 0://No signal level match. Use sum(i0) as denominator. Linear.
			scale1=1.f;
			scale2=cuwfs[iwfs].i0sum();
			break;
		case 1://Use instantaneous intensity of each sa
			break;
		case 2://Use averaged instantaneous intensity.
			scale1=cursum(ints.M(), stream)/PR(powfs[ipowfs].saasum, wfsind);
			scale2=cupowfs[ipowfs].saa.R(wfsind)();
			break;
		default:
			error("Invalid sigmatch\n");
		}
		Real* srot=parms->powfs[ipowfs].radpix?cuwfs[iwfs].srot():NULL;
		Real* cogmask=cuwfs[iwfs].cogmask();
		if(cogmask){
			wcog_do<<<nsa, dim3(pixpsax, pixpsay), 0, stream>>>
			(gradcalc, ints[0], scale1, scale2,
				pixpsax, pixpsay, pixthetax, pixthetay, nsa,
				cogmask, cogoff, srot);
		}else{
			tcog_do<<<nsa, dim3(pixpsax, pixpsay), 0, stream>>>
			(gradcalc, ints[0], scale1, scale2,
				pixpsax, pixpsay, pixthetax, pixthetay, nsa,
				cogthres, cogoff, srot);
		}
	}
	break;
	default://Use CPU version
		cp2cpu(&simu->ints->p[iwfs], ints, stream);//this already syncs stream.
		info_once("Calling shwfs_grad CPU version for phytype %d\n", parms->powfs[ipowfs].phytype_sim);
		shwfs_grad(&simu->gradcl->p[iwfs], P(simu->ints->p[iwfs]),
			parms, powfs, iwfs, parms->powfs[ipowfs].phytype_sim);
	}
	CUDA_CHECK_ERROR;
}
/**
   Ray tracing and gradient computation for WFS. \todo Expand to do gradients in GPU without transfering
   data back to CPU.
*/
void *gpu_wfsgrad_queue(thread_t* info){
	sim_t* simu=(sim_t*)info->data;
	const parms_t *parms=simu->parms;
	const powfs_t *powfs=simu->powfs;
	const recon_t *recon=simu->recon;
	const int CL=parms->sim.closeloop;
	const int isim=simu->wfsisim;
	for(int iwfs=info->start; iwfs<info->end; iwfs++){
		const int ipowfs=parms->wfs[iwfs].powfs;
		if(isim<parms->powfs[ipowfs].step) continue;
		gpu_set(cuglobal->wfsgpu[iwfs]);
		Array<cupowfs_t>& cupowfs=cudata->powfs;
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		/*output */
		/*The following are truly constants for this powfs */
		const int imoao=parms->powfs[ipowfs].moao;
		const int nsa=powfs[ipowfs].saloc->nloc;
		const int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
		const int dtrat=parms->powfs[ipowfs].dtrat;
		const int save_gradgeom=parms->save.gradgeom->p[iwfs];
		const int save_opd=parms->save.wfsopd->p[iwfs];
		const int save_ints=parms->save.ints->p[iwfs];
		const int noisy=parms->powfs[ipowfs].noisy;
		/*The following depends on isim */
		const int dtrat_output=((isim+1)%dtrat==0);
		//usephy is always true with dither is set
		const int do_phy=(parms->powfs[ipowfs].usephy&&isim>=parms->powfs[ipowfs].phystep);
		const int do_pistatout=parms->powfs[ipowfs].pistatout&&isim>=parms->powfs[ipowfs].pistatstart;
		const int do_geom=!do_phy||save_gradgeom||do_pistatout;
		const Real thetax=parms->wfs[iwfs].thetax;
		const Real thetay=parms->wfs[iwfs].thetay;
		const Real misregx=parms->powfs[ipowfs].type==WFS_SH?parms->wfs[iwfs].misregx:0;
		const Real misregy=parms->powfs[ipowfs].type==WFS_SH?parms->wfs[iwfs].misregy:0;
		Real2* loc=cupowfs[ipowfs].loc();
		/*Out to host for now. \todo : keep grad in device when do reconstruction on device. */
		stream_t& stream=cuwfs[iwfs].stream;
		curmat phiout=cuwfs[iwfs].phiout;
		curmat gradacc=cuwfs[iwfs].gradacc;
		curmat gradcalc=cuwfs[iwfs].gradcalc;
		curmat gradref;
		ctoc_init(30);
		if(isim%dtrat==0){
			cuzero(cuwfs[iwfs].ints, stream);
			cuzero(cuwfs[iwfs].gradacc, stream);
		}
		const int nhs=parms->powfs[ipowfs].llt?parms->powfs[ipowfs].llt->nhs:1;//number of sublayer
		const Real dhs=parms->powfs[ipowfs].llt?parms->powfs[ipowfs].llt->dhs/nhs:0;//spacing of sublayer
		for(int ihs=0; ihs<nhs; ihs++){
			const Real hs=(nhs>1?(ihs-(nhs-1)*0.5):0)*dhs+parms->wfs[iwfs].hs;
			const Real hc=nhs>1?(parms->wfs[iwfs].hc*(1.-hs/parms->wfs[iwfs].hs)):0;//effective hc
			if(cuwfs[iwfs].opdadd){ /*copy to phiout. */
				Copy(phiout, cuwfs[iwfs].opdadd, stream);
			} else{
				cuzero(phiout, stream);
			}
			if(simu->atm&&((!parms->sim.idealwfs&&!parms->powfs[ipowfs].lo)
							||(!parms->sim.wfsalias&&parms->powfs[ipowfs].lo))){
				atm2loc(phiout, cuwfs[iwfs].loc_tel, hs, hc, thetax, thetay, 
					misregx, misregy, parms->sim.dt, isim, 1, stream);
			}
			if(!parms->powfs[ipowfs].lo&&(parms->sim.idealwfs||parms->sim.wfsalias)){
				Real alpha=parms->sim.idealwfs?1:-1;
				if(parms->sim.idealwfs==2||parms->sim.wfsalias==2){
					dmat* opd=dnew(phiout.Nx(), 1);
					add2cpu(&opd, 0, phiout, 1, stream, 0);
					wfs_ideal_atm(simu, opd, iwfs, alpha);
					cp2gpu(phiout, opd);
					dfree(opd);
				} else{
					mapcell2loc(phiout, cuwfs[iwfs].loc_dm, cudata->dmproj, 
						hs, hc, thetax, thetay, 0, 0, alpha, stream);
				}
			}
			if(simu->telws){
				Real tt=simu->telws->p[isim];
				Real angle=simu->winddir?simu->winddir->p[0]:0;
				curaddptt(phiout, loc, 0, tt*cosf(angle), tt*sinf(angle), stream);
			}
			if(save_opd){
				zfarr_push_scale(simu->save->wfsopdol[iwfs], isim, phiout, 1, stream);
			}
			if(CL){
				wait_dmreal(simu, isim);
				mapcell2loc(phiout, cuwfs[iwfs].loc_dm, cudata->dmreal,
					hs, hc, thetax, thetay, parms->wfs[iwfs].misregx, parms->wfs[iwfs].misregy, -1, stream);
				Real ttx=0, tty=0;
				if(simu->ttmreal){
					ttx+=simu->ttmreal->p[0];
					tty+=simu->ttmreal->p[1];
				}
				if(simu->fsmreal&&PN(simu->fsmreal, iwfs)&&!powfs[ipowfs].llt){
					ttx+=simu->fsmreal->p[iwfs]->p[0];
					tty+=simu->fsmreal->p[iwfs]->p[1];
				}
				if(ttx||tty){
					curaddptt(phiout, loc, 0, -ttx, -tty, stream);
				}
			}

			if(parms->tomo.ahst_idealngs==1&&parms->powfs[ipowfs].lo){
				const real* cleNGSm=simu->cleNGSm->p+isim*recon->ngsmod->nmod;
				ngsmod2loc(phiout, cupowfs[ipowfs].loc(), recon->ngsmod, cleNGSm,
					parms->wfs[iwfs].thetax, parms->wfs[iwfs].thetay,
					-1, stream);
			}
			if(imoao>-1){
				mapcell2loc(phiout, cuwfs[iwfs].loc_dm, cudata->dm_wfs[iwfs], 
					INFINITY, hc, 0, 0, 0, 0, -1, stream);
			}

			Real focus=(Real)wfsfocusadj(simu, iwfs);
			if(Z(fabs)(focus)>1e-20){
				const int nloc=cupowfs[ipowfs].loc.Nloc();
				add_focus_do<<<DIM(nloc, 256), 0, stream>>>(phiout, loc, nloc, focus);
			}

			if(cupowfs[ipowfs].fieldstop){
				if(parms->powfs[ipowfs].nwvl>1){
					error("Implement broadband case\n");
				}
				cu_fieldstop(phiout, cuwfs[iwfs].amp, cupowfs[ipowfs].embed[0], cupowfs[ipowfs].nembed[0],
					cupowfs[ipowfs].fieldstop.R(0,wfsind), parms->powfs[ipowfs].wvl->p[0], cuwfs[iwfs].plan_fs, stream);
			}
			if(save_opd){
				zfarr_push_scale(simu->save->wfsopd[iwfs], isim, phiout, 1, stream);
			}
			if(parms->plot.run&&isim%parms->plot.run==0){
				const dmat* amp=PR(powfs[ipowfs].amp, wfsind);
				drawopdamp_gpu("Opdwfs", powfs[ipowfs].loc, phiout, stream, amp, 0,
					"WFS OPD", "x (m)", "y (m)", "WFS %2d", iwfs);
			}
			ctoc("opd");
			if(parms->powfs[ipowfs].type==WFS_PY){
				if(nhs>1) error("Please implement\n");
				CUDA_CHECK_ERROR;
				pywfs_ints(cuwfs[iwfs].ints[0], phiout, cuwfs[iwfs], parms->wfs[iwfs].sigsim);
				ctoc("pywfs");
				CUDA_CHECK_ERROR;
			} else{
				if(do_geom){
					real ratio;
					if(do_pistatout&&(dtrat>1||nhs>1)){//compute each
						gradref=gradcalc;
						cuzero(gradcalc, stream);
						ratio=1;
					} else{
						gradref=gradacc;
						ratio=1.f/(dtrat*nhs);
					}

					if(parms->powfs[ipowfs].gtype_sim==GTYPE_Z){
						cuztilt(gradref, phiout,
							cupowfs[ipowfs].pts.Nloc(),
							cupowfs[ipowfs].pts.Dxsa(),
							cupowfs[ipowfs].pts.Nxsa(), cuwfs[iwfs].imcc,
							cupowfs[ipowfs].pts(), cuwfs[iwfs].amp, ratio, stream);
					} else{
						cuspmul(gradref, cuwfs[iwfs].GS0, phiout, 1, 'n', ratio, stream);
					}
					if(gradacc()!=gradref()){
						Add(gradacc, (Real)1, gradref, (Real)1.f/(dtrat*nhs), stream);
					}
				}
				if(parms->powfs[ipowfs].psfout){
					zfarr_push_scale(simu->save->ztiltout[iwfs], isim, gradcalc, 1, stream);
				}
				if(do_phy||parms->powfs[ipowfs].psfout||do_pistatout){/*physical optics */
					if(nhs>1) error("Please implement\n");
					CUDA_CHECK_ERROR;
					wfsints(simu, phiout, gradref, iwfs, isim);
					CUDA_CHECK_ERROR;
					ctoc("shwfs");
				}/*do phy */
			}
			if(dtrat_output && (ihs+1)==nhs){
				Real rne=0, bkgrnd=0;
				if(do_phy){
					curcell &ints=cuwfs[iwfs].ints;	/*signal level was already multiplied in ints. */
					const int totpix=(powfs[ipowfs].pywfs)?powfs[ipowfs].pywfs->cfg->nside:(ints[0].N());//PyWFs and SHWFS
					if(save_ints){
						zfarr_push_scale(simu->save->intsnf[iwfs], isim, ints, 1, stream);
					}
					if(noisy){
						if(parms->save.gradnf->p[iwfs]){
							if(parms->powfs[ipowfs].type==WFS_PY){//PWFS
								pywfs_grad(gradcalc, cuwfs[iwfs].ints[0], cupowfs[ipowfs].saa.R(wfsind),
									cuwfs[iwfs].isum, cupowfs[ipowfs].pyoff, powfs[ipowfs].pywfs, stream);
							} else{
								shwfs_grad(gradcalc, cuwfs[iwfs].ints, cuwfs, cupowfs, parms, powfs, simu, iwfs, ipowfs, stream);
							}
							if(parms->powfs[ipowfs].phytype_sim<3){
								zfarr_push_scale(simu->save->gradnf[iwfs], isim, gradcalc, 1, stream);
							} else{//CPU version
								zfarr_push(simu->save->gradnf[iwfs], isim, simu->gradcl->p[iwfs]);
							}
						}
						rne=parms->powfs[ipowfs].rne;
						bkgrnd=parms->powfs[ipowfs].bkgrnd*dtrat;
						addnoise_do<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt, 0, stream>>>
							(ints[0], nsa, totpix, bkgrnd, bkgrnd*parms->powfs[ipowfs].bkgrndc,
								cuwfs[iwfs].bkgrnd2(), cuwfs[iwfs].bkgrnd2c(),
								cuwfs[iwfs].qe, rne, cuwfs[iwfs].custat);
						ctoc("noise");
						if(save_ints){
							zfarr_push_scale(simu->save->intsny[iwfs], isim, ints, 1, stream);
						}
					}
					if(parms->powfs[ipowfs].i0save){
						curcelladd(cuwfs[iwfs].intsout, 1, ints, 1, stream);
					}
					if(abs(parms->powfs[ipowfs].dither)==1
						&&isim>=parms->powfs[ipowfs].dither_ogskip
						&&parms->powfs[ipowfs].type==WFS_SH
						&&(parms->powfs[ipowfs].dither==-1||parms->powfs[ipowfs].phytype_sim2==PTYPE_MF)){
						int npll=parms->powfs[ipowfs].dither_pllrat;
						if(parms->powfs[ipowfs].dither==1){
							real cs=0, ss=0;
							dither_position(&cs, &ss, parms->powfs[ipowfs].alfsm, parms->powfs[ipowfs].dtrat,
								parms->powfs[ipowfs].dither_npoint, isim, simu->dither[iwfs]->deltam);
							cuwfs[iwfs].dither.acc(simu->dither[iwfs], ints, cs, ss, npll, stream);
						}else{//just accumulate i0
							cuwfs[iwfs].dither.acc_i0(simu->dither[iwfs], ints, npll, stream);
						}
						ctoc("dither");
					}

					if(parms->powfs[ipowfs].type==WFS_PY){
						pywfs_grad(gradcalc, cuwfs[iwfs].ints[0], cupowfs[ipowfs].saa.R(wfsind),
							cuwfs[iwfs].isum, cupowfs[ipowfs].pyoff, powfs[ipowfs].pywfs, stream);
						//cuwrite(gradcalc, stream, "gradcalc"); exit(0);
					} else{
						shwfs_grad(gradcalc, cuwfs[iwfs].ints, cuwfs, cupowfs, parms, powfs, simu, iwfs, ipowfs, stream);
					}
					ctoc("grad");

				} else{//geometric
					if(noisy){
						if(parms->save.gradnf->p[iwfs]){
							zfarr_push_scale(simu->save->gradnf[iwfs], isim, gradacc, 1, stream);
						}
						if(!parms->powfs[ipowfs].usephy){//do not add noise for presimulation to physical optics
							add_geom_noise_do<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt, 0, stream>>>
								(gradacc, cuwfs[iwfs].neasim, nsa, cuwfs[iwfs].custat);
							ctoc("geom_noise");
						}
					}
				}
			}/*dtrat_output */
			//info("thread %ld gpu %d iwfs %d queued\n", thread_id(), cudata->igpu, iwfs);
			ctoc_final("wfs %d", iwfs);

			/*{//this replaces gpu_wfsgrad_sync
				//This does not work. memcpy is not allowed in callback
				struct wfsgrad_callback_t* tmp=mycalloc(1, struct wfsgrad_callback_t);
				tmp->simu=simu;
				tmp->iwfs=iwfs;
				tmp->op=2;
				hipStreamAddCallback(stream, wfsgrad_callback, (void*)tmp, 0);
			}*/
			CUDA_CHECK_ERROR;
		}//for ihs
	}//for iwfs
	return NULL;
}

void gpu_wfsgrad_sync(sim_t* simu, int iwfs){
	const parms_t* parms=simu->parms;
	gpu_set(cuglobal->wfsgpu[iwfs]);
	Array<cuwfs_t>& cuwfs=cuglobal->wfs;
	stream_t& stream=cuwfs[iwfs].stream;
	CUDA_SYNC_STREAM;
	post_dmreal(simu);
	const int isim=simu->wfsisim;
	const int ipowfs=parms->wfs[iwfs].powfs;
	const int dtrat=parms->powfs[ipowfs].dtrat;
	const int dtrat_output=((isim+1)%dtrat==0);
	if(dtrat_output){
		const int save_gradgeom=parms->save.gradgeom->p[iwfs];
		const int do_phy=(parms->powfs[ipowfs].usephy&&isim>=parms->powfs[ipowfs].phystep);
		if(do_phy){
			if(parms->powfs[ipowfs].phytype_sim<3){//3 is handled in cpu.
				cp2cpu(&simu->gradcl->p[iwfs], cuwfs[iwfs].gradcalc, stream);
			}
			if(save_gradgeom){//also do geom grad during phy grad sims
				zfarr_push_scale(simu->save->gradgeom[iwfs], isim, cuwfs[iwfs].gradacc, 1, stream);
			}
			if((parms->plot.run&&isim%parms->plot.run==0&&draw_current("Ints", NULL))
			||(parms->powfs[ipowfs].lo && parms->recon.petal)){// && parms->powfs[ipowfs].lo){
				cp2cpu(&simu->ints->p[iwfs], cuwfs[iwfs].ints, stream);
			}
		} else{
			cp2cpu(&simu->gradcl->p[iwfs], cuwfs[iwfs].gradacc, stream);
		}
	}
}
void gpu_save_pistat(sim_t* simu){
	const parms_t* parms=simu->parms;
	const int isim=simu->wfsisim;
	for(int iwfs=0; iwfs<simu->parms->nwfs; iwfs++){
		gpu_set(cuglobal->wfsgpu[iwfs]);
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		const int ipowfs=simu->parms->wfs[iwfs].powfs;
		stream_t& stream=cuwfs[iwfs].stream;
		if(parms->powfs[ipowfs].pistatout){
			int nstep=(isim+1-parms->powfs[ipowfs].pistatstart);
			if(nstep>0){
				curcell tmp=cuwfs[iwfs].pistatout;
				curcellscale(tmp, 1.f/(Real)nstep, stream);
				if(parms->sim.skysim){
					cuwrite(tmp, stream, "%s/pistat/pistat_seed%d_sa%d_x%g_y%g.bin",
						dirskysim, simu->seed,
						parms->powfs[ipowfs].order,
						parms->wfs[iwfs].thetax*206265,
						parms->wfs[iwfs].thetay*206265);
				} else{
					cuwrite(tmp, stream, "pistat_seed%d_wfs%d.bin", simu->seed, iwfs);
				}
				curcellscale(tmp, nstep, stream);
			}
		}

		if(parms->powfs[ipowfs].i0save){
			const int dtrat=parms->powfs[ipowfs].dtrat;
			const int nstep=(isim+1-parms->powfs[ipowfs].phystep)/dtrat;
			if(nstep>0){
				curcell tmp=cuwfs[iwfs].intsout;
				curcellscale(tmp, 1.f/(Real)nstep, stream);
				cuwrite(tmp, stream, "ints_%d_wfs%d.bin", simu->seed, iwfs);
				curcellscale(tmp, nstep, stream);
			}
		}
	}
}
