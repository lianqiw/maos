#include "hip/hip_runtime.h"
/*
  Copyright 2009-2025 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <hiprand/hiprand_kernel.h>
#include "../math/cumath.h"
#include "accphi.h"
#include "wfs.h"
#include "cudata.h"

static void etf2gpu(cucmat& cuetf, etf_t* etf, int icol){
	cmat* etfm=ccell_col(etf->etf, icol);
	cp2gpu(cuetf, etfm);
	cfree(etfm);
}
/**
   Initialize or update etf.
*/
void gpu_wfsgrad_update_etf(const parms_t* parms, const powfs_t* powfs, int ipowfs){
	const int* wfsgpu=cuglobal->wfsgpu();
	TIC;tic;
	if(parms->powfs[ipowfs].usephy&&parms->powfs[ipowfs].llt){
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		const int nwvl=parms->powfs[ipowfs].nwvl;
		const int iwfs0=parms->powfs[ipowfs].wfs->p[0];
		for(int jwfs=0; jwfs<parms->powfs[ipowfs].nwfs; jwfs++){
			int iwfs=P(parms->powfs[ipowfs].wfs, jwfs);
			gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
			for(int iwvl=0; iwvl<nwvl; iwvl++){
				int skip_1=0;
				if(powfs[ipowfs].etfsim2){
					if(fabs(cuwfs[iwfs].dtf[iwvl].etf[1].hs-powfs[ipowfs].etfsim[iwvl].hs)<1
					&&cuwfs[iwfs].dtf[iwvl].etf[1].icol==powfs[ipowfs].etfsim[iwvl].icol){
						//dbg("Reuse etf2 as etf for next step for wfs %d\n", iwfs);
						skip_1=1;
						cucmat tmp=cuwfs[iwfs].dtf[iwvl].etf[0].etf;
						cuwfs[iwfs].dtf[iwvl].etf[0].etf=cuwfs[iwfs].dtf[iwvl].etf[1].etf;
						cuwfs[iwfs].dtf[iwvl].etf[1].etf=tmp;
					}
				}
				if(jwfs==0||parms->powfs[ipowfs].llt->nllt>1){
					int kwfs=parms->powfs[ipowfs].llt->nllt>1?jwfs:0;
					if(powfs[ipowfs].etfsim && !skip_1){
						//dbg("Copy etf from wfs %d from CPU to GPU\n", iwfs);
						etf2gpu(cuwfs[iwfs].dtf[iwvl].etf[0].etf, &powfs[ipowfs].etfsim[iwvl], kwfs);
					}
					if(powfs[ipowfs].etfsim2){
						//dbg("Copy etf2 from wfs %d from CPU to GPU\n", iwfs);
						etf2gpu(cuwfs[iwfs].dtf[iwvl].etf[1].etf, &powfs[ipowfs].etfsim2[iwvl], kwfs);
					}
				} else {//copy across GPU
					int iwfs2;
					for(iwfs2=iwfs0; iwfs2<iwfs; iwfs2++){
						if(wfsgpu[iwfs2]==wfsgpu[iwfs]){
							break;
						}
					}
					if(iwfs2==iwfs){
						if(powfs[ipowfs].etfsim&&!skip_1){
							//dbg("Copy etf from wfs %d to %d within GPU\n", iwfs0, iwfs);
							cuwfs[iwfs].dtf[iwvl].etf[0].etf.Copy(cuwfs[iwfs0].dtf[iwvl].etf[0].etf);
						}
						if(powfs[ipowfs].etfsim2){
							//dbg("Copy etf2 from wfs %d to %d within GPU\n", iwfs0, iwfs);
							cuwfs[iwfs].dtf[iwvl].etf[1].etf.Copy(cuwfs[iwfs0].dtf[iwvl].etf[1].etf);
						}
					}else{//reference
						if(powfs[ipowfs].etfsim&&!skip_1){
							//dbg("Reference etf from wfs %d to %d within GPU\n", iwfs2, iwfs);
							cuwfs[iwfs].dtf[iwvl].etf[0].etf=cuwfs[iwfs2].dtf[iwvl].etf[0].etf;
						}
						if(powfs[ipowfs].etfsim2){
							//dbg("Reference etf2 from wfs %d to %d within GPU\n", iwfs2, iwfs);
							cuwfs[iwfs].dtf[iwvl].etf[1].etf=cuwfs[iwfs2].dtf[iwvl].etf[1].etf;
						}
					}
				}
				if(powfs[ipowfs].etfsim){
					cuwfs[iwfs].dtf[iwvl].etf[0].icol=powfs[ipowfs].etfsim[iwvl].icol;
					cuwfs[iwfs].dtf[iwvl].etf[0].hs=powfs[ipowfs].etfsim[iwvl].hs;
				}
				if(powfs[ipowfs].etfsim2){
					cuwfs[iwfs].dtf[iwvl].etf[1].icol=powfs[ipowfs].etfsim2[iwvl].icol;
					cuwfs[iwfs].dtf[iwvl].etf[1].hs=powfs[ipowfs].etfsim2[iwvl].hs;
				}
			}
		}
	}
	toc2("gpu_wfsgrad_update_etf");
}
/**
   Initialize or update mtched filter, cogmask, and i0sum, i0sumsum
*/
void gpu_wfsgrad_update_ref(const parms_t* parms, const powfs_t* powfs, int ipowfs){
	const int* wfsgpu=cuglobal->wfsgpu();
	Array<cuwfs_t>& cuwfs=cuglobal->wfs;
	if(parms->powfs[ipowfs].usephy&&powfs[ipowfs].intstat){
		dbg("powfs%d: updating matched filter in GPU\n", ipowfs);
		const int iwfs0=parms->powfs[ipowfs].wfs->p[0];
		const int nsa=powfs[ipowfs].saloc->nloc;
		const int multi=(parms->powfs[ipowfs].phytype_sim==PTYPE_MF&&NY(powfs[ipowfs].intstat->mtche)>1)
		||parms->powfs[ipowfs].phytype_sim==PTYPE_COG&&NY(powfs[ipowfs].intstat->cogmask)>1;
		for(int jwfs=0; jwfs<parms->powfs[ipowfs].nwfs; jwfs++){
			int iwfs=P(parms->powfs[ipowfs].wfs, jwfs);
			gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
			int iwfs2;
			for(iwfs2=iwfs0; iwfs2<iwfs; iwfs2++){
				if(wfsgpu[iwfs2]==wfsgpu[iwfs]){
					break;
				}
			}
			if(multi||iwfs2==iwfs){
				const int icol=multi?jwfs:0;
				if(parms->powfs[ipowfs].phytype_sim==PTYPE_MF){//matched filter
					dmat* mtche=dcell_col(powfs[ipowfs].intstat->mtche, icol);
					//Delete old referenced values.
					cuwfs[iwfs].mtche.deinit();
					cp2gpu(cuwfs[iwfs].mtche, mtche);
					dfree(mtche);
				}
				if(parms->powfs[ipowfs].phytype_sim==PTYPE_COG && powfs[ipowfs].intstat->cogmask){//matched filter
					dmat* cogmask=dcell_col(powfs[ipowfs].intstat->cogmask, icol);
					//Delete old referenced values.
					cuwfs[iwfs].cogmask.deinit();
					cp2gpu(cuwfs[iwfs].cogmask, cogmask);
					dfree(cogmask);
				}
				if(powfs[ipowfs].intstat->i0sum){
					//Delete old referenced values.
					cuwfs[iwfs].i0sum.deinit();
					cp2gpu(cuwfs[iwfs].i0sum, &PR(powfs[ipowfs].intstat->i0sum, 0, jwfs), nsa, 1);
					cuwfs[iwfs].i0sumsum=PR(powfs[ipowfs].intstat->i0sumsum, jwfs, 0);
				}
			} else{
				cuwfs[iwfs].mtche=cuwfs[iwfs2].mtche;
				cuwfs[iwfs].cogmask=cuwfs[iwfs2].cogmask;
				cuwfs[iwfs].i0sum=cuwfs[iwfs2].i0sum;
				cuwfs[iwfs].i0sumsum=cuwfs[iwfs2].i0sumsum;
			}
		}
	}
}
/**
   Initialize other arrays
*/
void gpu_wfsgrad_init(const parms_t* parms, const powfs_t* powfs){
	const int* wfsgpu=cuglobal->wfsgpu();
	cuglobal->wfs.init(parms->nwfs, 1);
	for(int im=0; im<NGPU; im++){
		gpu_set(im);
		cudata->powfs.init(parms->npowfs, 1);
		Array<cupowfs_t>& cupowfs=cudata->powfs;

		/* Setup information that are same for wfs in each powfs*/
		for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
			if(parms->powfs[ipowfs].nwfs==0) continue;
			loc_t* loc=powfs[ipowfs].loc;
			cupowfs[ipowfs].loc=culoc_t(loc);
			if(parms->powfs[ipowfs].type==WFS_PY){//only for pywfs
				cupowfs[ipowfs].saloc=culoc_t(powfs[ipowfs].saloc);
			}
			pts_t* pts=powfs[ipowfs].pts;
			if(pts) cupowfs[ipowfs].pts=cupts_t(pts);

			if(powfs[ipowfs].llt&&parms->powfs[ipowfs].trs){
				pts=powfs[ipowfs].llt->pts;
				loc=powfs[ipowfs].llt->loc;
				cupowfs[ipowfs].llt.pts=cupts_t(pts);
				cupowfs[ipowfs].llt.loc=culoc_t(loc);
			}
			/*cupowfs[ipowfs].skip=parms->powfs[ipowfs].skip; */
			locfft_t** plocfft=0;
			int nlocfft=0;
			if(powfs[ipowfs].pywfs){
				plocfft=&powfs[ipowfs].pywfs->locfft;
				nlocfft=1;
			} else if(powfs[ipowfs].fieldstop){
				plocfft=powfs[ipowfs].fieldstop;
				nlocfft=PN(powfs[ipowfs].amp);
			}
			if(plocfft){
				const int nwvl=parms->powfs[ipowfs].nwvl;
				cupowfs[ipowfs].embed.init(nwvl, 1);
				cupowfs[ipowfs].nembed.init(nwvl, 1);
				if((*plocfft)->fieldmask){
					cupowfs[ipowfs].fieldstop=curcell(nwvl, nlocfft);
				}
				for(int iwvl=0; iwvl<nwvl; iwvl++){
					cp2gpu(cupowfs[ipowfs].embed[iwvl], P(plocfft[0]->embed->p[iwvl]), powfs[ipowfs].loc->nloc, 1);
					cupowfs[ipowfs].nembed[iwvl]=plocfft[0]->nembed->p[iwvl];
					if(plocfft[0]->fieldmask){
						if(nwvl>1){
							error("nwvl>1 case is not supported\n");
						}
						for(int iwfs=0; iwfs<PN(powfs[ipowfs].amp); iwfs++){
							cp2gpu(cupowfs[ipowfs].fieldstop(iwvl,iwfs), plocfft[iwfs]->fieldmask->p[iwvl]);
						}
					}
				}
			}
			cp2gpu(cupowfs[ipowfs].saa, powfs[ipowfs].saa);
			cp2gpu(cupowfs[ipowfs].pixoffx, powfs[ipowfs].pixoffx);
			cp2gpu(cupowfs[ipowfs].pixoffy, powfs[ipowfs].pixoffy);
			if(powfs[ipowfs].pywfs){
				powfs[ipowfs].pywfs->gpu=1;
				cupowfs[ipowfs].pywfs=powfs[ipowfs].pywfs;
				cp2gpu(cupowfs[ipowfs].pyramid, powfs[ipowfs].pywfs->pyramid);
				cp2gpu(cupowfs[ipowfs].pynominal, powfs[ipowfs].pywfs->nominal);
				cp2gpu(cupowfs[ipowfs].pyoff, powfs[ipowfs].pywfs->gradoff);
				if(powfs[ipowfs].pywfs->msaloc){
					cupowfs[ipowfs].msaloc.init(powfs[ipowfs].pywfs->msaloc->nx, 1);
					for(int i=0; i<powfs[ipowfs].pywfs->msaloc->nx;i++){
						cupowfs[ipowfs].msaloc[i]=culoc_t(powfs[ipowfs].pywfs->msaloc->p[i]);
					}
				}
			}
		}
	}

	/* setup information that maybe different for wfs in same powfs due to
	   misregistration or NCPA.*/
	for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
		const int ipowfs=parms->wfs[iwfs].powfs;
		gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
		Array<cupowfs_t>& cupowfs=cudata->powfs;
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		cuwfs[iwfs].stream.reset();//Recreate streams in current GPU.
		cuwfs[iwfs].powfs=&cupowfs[ipowfs];
		const int nsa=powfs[ipowfs].saloc->nloc;
		const int nwvl=parms->powfs[ipowfs].nwvl;
		const int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
		const int iwfs0=parms->powfs[ipowfs].wfs->p[0];
		const int ndm=parms->ndm;
		/*imcc for ztilt. */
		cuwfs[iwfs].loc_dm.init(ndm, 1);
		for(int idm=0; idm<ndm; idm++){
			if(powfs[ipowfs].loc_dm){
				cuwfs[iwfs].loc_dm[idm]=culoc_t(P(powfs[ipowfs].loc_dm, wfsind, idm));
			}else if(powfs[ipowfs].loc_tel){
				cuwfs[iwfs].loc_dm[idm]=culoc_t(P(powfs[ipowfs].loc_tel, wfsind));
			} else{
				cuwfs[iwfs].loc_dm[idm]=culoc_t(powfs[ipowfs].loc);
			}
		}
		if(powfs[ipowfs].loc_tel){
			cuwfs[iwfs].loc_tel=culoc_t(powfs[ipowfs].loc_tel->p[wfsind]);
		} else{
			cuwfs[iwfs].loc_tel=culoc_t(powfs[ipowfs].loc);
		}
		cuwfs[iwfs].phiout=curmat(powfs[ipowfs].loc->nloc, 1);
		if(cupowfs[ipowfs].nembed){
			DO(hipfftPlan2d(&cuwfs[iwfs].plan_fs, cupowfs[ipowfs].nembed[0], cupowfs[ipowfs].nembed[0], FFT_T_C2C));
			DO(hipfftSetStream(cuwfs[iwfs].plan_fs, cuwfs[iwfs].stream));
		}
		if(powfs[ipowfs].saimcc){
			if(powfs[ipowfs].saimcc->nx>1||wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
				int icol=powfs[ipowfs].saimcc->nx>1?wfsind:0;
				dmat* imcc=dcell_col(powfs[ipowfs].saimcc->p[icol], 0);
				cp2gpu(cuwfs[iwfs].imcc, imcc);
				dfree(imcc);
			} else{
				cuwfs[iwfs].imcc=cuwfs[iwfs0].imcc;
			}
		}
		/*GS0 for gtilt. */
		if(powfs[ipowfs].GS0){
			cuwfs[iwfs].GS0=cusp(powfs[ipowfs].GS0->p[powfs[ipowfs].GS0->nx>1?wfsind:0]);
		}
		/*wfs amplitude map on loc */
		cp2gpu(cuwfs[iwfs].amp, PR(powfs[ipowfs].amp, wfsind));
		if(powfs[ipowfs].neasim){//neasim is the LL' decomposition of nea
			dmat* neasim=PR(powfs[ipowfs].neasim, wfsind, 0);
			if(neasim) cp2gpu(cuwfs[iwfs].neasim, neasim);
		}
		/* * Now start physical optics setup * */
		if(parms->powfs[ipowfs].type==WFS_PY){//Pyramid WFS
			cuwfs[iwfs].pywvf=cuccell(nwvl, 1);
			for(int iwvl=0; iwvl<nwvl; iwvl++){
				cuwfs[iwfs].pywvf[iwvl]=cucmat(cupowfs[ipowfs].nembed[iwvl], cupowfs[ipowfs].nembed[iwvl]);
			}
			const int nxotf=cupowfs[ipowfs].pyramid[0].Nx();;
			const int nyotf=cupowfs[ipowfs].pyramid[0].Ny();
			cuwfs[iwfs].pyotf=cucmat(nxotf, nyotf);
			cuwfs[iwfs].pypsf=curmat(nxotf, nyotf);
			DO(hipfftPlan2d(&cuwfs[iwfs].plan_py, nyotf, nxotf, FFT_T_C2C));
			hipfftSetStream(cuwfs[iwfs].plan_py, cuwfs[iwfs].stream);
			cuwfs[iwfs].isum=curmat(1, 1);
		} else if(parms->powfs[ipowfs].usephy||parms->powfs[ipowfs].psfout||parms->powfs[ipowfs].pistatout){
			/*If there is llt. */
			if(powfs[ipowfs].llt&&parms->powfs[ipowfs].trs){
				if(powfs[ipowfs].llt->ncpa){
					if(powfs[ipowfs].llt->ncpa->nx>1||wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
						cp2gpu(cuwfs[iwfs].lltncpa,
							powfs[ipowfs].llt->ncpa->p[powfs[ipowfs].llt->ncpa->nx>1?wfsind:0]);
					} else{
						cuwfs[iwfs].lltncpa=cuwfs[iwfs0].lltncpa;
					}
				}
				if(wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
					cp2gpu(cuwfs[iwfs].lltimcc, powfs[ipowfs].llt->imcc->p[0]);
					cp2gpu(cuwfs[iwfs].lltamp, powfs[ipowfs].llt->amp);
				} else{
					cuwfs[iwfs].lltimcc=cuwfs[iwfs0].lltimcc;
					cuwfs[iwfs].lltamp=cuwfs[iwfs0].lltamp;
				}
			}
			/*CUFFTW is row major. */
			int nwvf=powfs[ipowfs].pts->nxsa*parms->powfs[ipowfs].embfac;/*size of fft */
			int nwvf2[2]={nwvf, nwvf};
			const int notfx=powfs[ipowfs].notfx;
			const int notfy=powfs[ipowfs].notfy;
			int notf2[2]={notfy, notfx};
			/*limit the number of subapertures in each batch to less than 1024
			  to save memory. The speed is actually a tiny bit faster for NFIRAOS.*/
			cuwfs[iwfs].msa=(nsa>1024)?((int)ceil((Real)nsa/(Real)(nsa/800))):nsa;
			if(hipfftPlanMany(&cuwfs[iwfs].plan_wvf, 2, nwvf2, NULL, 1, 0, NULL, 1, 0,
							 FFT_T_C2C, cuwfs[iwfs].msa)){
				error("CUFFT plan failed\n");
			}
			hipfftSetStream(cuwfs[iwfs].plan_wvf, cuwfs[iwfs].stream);

			if(notfx==nwvf&&notfx==notfy){
				cuwfs[iwfs].plan_psf=cuwfs[iwfs].plan_wvf;
			} else{
				if(hipfftPlanMany(&cuwfs[iwfs].plan_psf, 2, notf2, NULL, 1, 0, NULL, 1, 0, FFT_T_C2C, cuwfs[iwfs].msa)){
					error("CUFFT plan failed\n");
				}
				hipfftSetStream(cuwfs[iwfs].plan_psf, cuwfs[iwfs].stream);
			}
			if(parms->powfs[ipowfs].llt){
				int nlwvf=powfs[ipowfs].llt->pts->nxsa*parms->powfs[ipowfs].embfac;
				int nlwvf2[2]={nlwvf, nlwvf};
				cuwfs[iwfs].lltwvf=cucmat(nlwvf, nlwvf);
				if(hipfftPlanMany(&cuwfs[iwfs].lltplan_wvf, 2, nlwvf2, NULL, 1, 0, NULL, 1, 0, FFT_T_C2C, 1)){
					error("CUFFT plan failed\n");
				}
				hipfftSetStream(cuwfs[iwfs].lltplan_wvf, cuwfs[iwfs].stream);
				if(nlwvf!=nwvf){
					cuwfs[iwfs].lltotfc2=cucmat(nwvf, nwvf);
					if(hipfftPlanMany(&cuwfs[iwfs].lltplan_lotfc2, 2, nwvf2, NULL, 1, 0, NULL, 1, 0, FFT_T_C2C, 1)){
						error("CUFFT plan failed\n");
					}
					hipfftSetStream(cuwfs[iwfs].lltplan_lotfc2, cuwfs[iwfs].stream);
				}else{
					cuwfs[iwfs].lltplan_lotfc2=cuwfs[iwfs].lltplan_wvf;
				}
				if(nwvf!=notfx||notfx!=notfy){
					cuwfs[iwfs].lltotfc=cucmat(notfx, notfy);
					if(hipfftPlanMany(&cuwfs[iwfs].lltplan_lotfc, 2, notf2, NULL, 1, 0, NULL, 1, 0, FFT_T_C2C, 1)){
						error("CUFFT plan failed\n");
					}
					hipfftSetStream(cuwfs[iwfs].lltplan_lotfc, cuwfs[iwfs].stream);
				}else{
					cuwfs[iwfs].lltplan_lotfc=cuwfs[iwfs].lltplan_lotfc2;
				}
			}
			/*DTF. */
			if(parms->powfs[ipowfs].usephy){
				if((parms->powfs[ipowfs].llt&&parms->powfs[ipowfs].llt->nllt>1)||wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
					 /*Need one per wfs in this powfs, or the first wfs. */
					cuwfs[iwfs].dtf.init(nwvl, 1);
					for(int iwvl=0; iwvl<nwvl; iwvl++){
						if(!parms->powfs[ipowfs].llt){
							int icol=powfs[ipowfs].dtf[iwvl].nominal->ny>1?wfsind:0;
							dmat* nominal=dcell_col(powfs[ipowfs].dtf[iwvl].nominal, icol);
							cp2gpu(cuwfs[iwfs].dtf[iwvl].nominal, nominal);
							dfree(nominal);
						}
						//ETF moved to gpu_wfsgrad_update_etf();
					}/*for iwvl. */
					if(parms->powfs[ipowfs].llt){
						cp2gpu(cuwfs[iwfs].srot, PR(powfs[ipowfs].srot, wfsind, 0));
					}
				} else{
					cuwfs[iwfs].dtf=cuwfs[iwfs0].dtf;
					cuwfs[iwfs].srot=cuwfs[iwfs0].srot;
				}
				if(wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
					cp2gpu(cuwfs[iwfs].qe, parms->powfs[ipowfs].qe);
				} else{
					cuwfs[iwfs].qe=cuwfs[iwfs0].qe;
				}
				//Matched filter is separated with gpu_wfsgrad_upate_mtche();
			
				if(powfs[ipowfs].bkgrnd){
					if(powfs[ipowfs].bkgrnd->ny==1||wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
						int icol=(powfs[ipowfs].bkgrnd->ny==1?wfsind:0);
						dmat* bkgrnd=dcell_col(powfs[ipowfs].bkgrnd, icol);
						cp2gpu(cuwfs[iwfs].bkgrnd2, bkgrnd);
						dfree(bkgrnd);
					} else{
						cuwfs[iwfs].bkgrnd2=cuwfs[iwfs0].bkgrnd2;
					}
				}
				if(powfs[ipowfs].bkgrndc){
					if(powfs[ipowfs].bkgrndc->ny==1||wfsind==0||wfsgpu[iwfs]!=wfsgpu[iwfs0]){
						int icol=(powfs[ipowfs].bkgrndc->ny==1?wfsind:0);
						dmat* bkgrnd=dcell_col(powfs[ipowfs].bkgrndc, icol);
						cp2gpu(cuwfs[iwfs].bkgrnd2c, bkgrnd);
						dfree(bkgrnd);

					} else{
						cuwfs[iwfs].bkgrnd2c=cuwfs[iwfs0].bkgrnd2c;
					}
				}
				if(parms->powfs[ipowfs].dither==1){
					cuwfs[iwfs].dither=Dither_t(nsa, powfs[ipowfs].pixpsax, powfs[ipowfs].pixpsay, 1);
				} else if(parms->powfs[ipowfs].dither==-1){
					cuwfs[iwfs].dither=Dither_t(nsa, powfs[ipowfs].pixpsax, powfs[ipowfs].pixpsay, 0);
				}
			}
			const int msa=cuwfs[iwfs].msa;
			cuwfs[iwfs].wvf=cucmat(nwvf*nwvf, msa);
			if(nwvf!=notfx||notfx!=notfy){
				cuwfs[iwfs].psf=cucmat(notfx*notfy, msa);
			}
			/*if(notfx!=notf||notfy!=notf){
				cuwfs[iwfs].otf=cucmat(notfx*notfy, msa);//no longer used
			}*/
			if(parms->powfs[ipowfs].psfout){
				cuwfs[iwfs].wvfout=cuccell(nsa, nwvl, notfx/2+2, notfy/2+2);
				cuwfs[iwfs].psfout=cucmat(notfx*notfy, msa);
			}
			if(parms->powfs[ipowfs].pistatout){
				cuwfs[iwfs].psfstat=cucmat(notfx*notfy, msa);
			}
			if(powfs[ipowfs].llt&&parms->powfs[ipowfs].trs){
				int nlx=powfs[ipowfs].llt->pts->nxsa;
				int nlwvf=nlx*parms->powfs[ipowfs].embfac;
				cuwfs[iwfs].lltopd=curmat(nlx, nlx);
				if(parms->powfs[ipowfs].pistatout||parms->powfs[ipowfs].idealfsm){
					cuwfs[iwfs].lltg.init(2, 1);
				}
				if(nlwvf!=nwvf){
					cuwfs[iwfs].lltotfc2=cucmat(nwvf, nwvf);
				}
				if(nwvf!=notfx||notfx!=notfy){
					cuwfs[iwfs].lltotfc=cucmat(notfx, notfy);
				}
			}
		}/*if phy */
		CUDA_SYNC_DEVICE;
	}/*for iwfs */
	for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
		gpu_wfsgrad_update_etf(parms, powfs, ipowfs);
		gpu_wfsgrad_update_ref(parms, powfs, ipowfs);
	}
	gpu_print_mem("wfs init");
}
/*
void gpu_wfs_update_amp(const parms_t* parms, powfs_t* powfs){
	for(int ipowfs=0; ipowfs<parms->npowfs; ipowfs++){
		for(int wfsind=0; wfsind<parms->powfs[ipowfs].nwfs; wfsind++){
			int iwfs=P(parms->powfs[ipowfs].wfs, wfsind);
			cp2gpu(cuwfs[iwfs].amp, PR(powfs[ipowfs].amp, wfsind));
		}
	}
}*/

void gpu_wfs_init_sim(const parms_t* parms, powfs_t* powfs){
	int* wfsgpu=cuglobal->wfsgpu();
	for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
		gpu_set(wfsgpu[iwfs]);/*Only initialize WFS in assigned GPU. */
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		int ipowfs=parms->wfs[iwfs].powfs;
		int nsa=powfs[ipowfs].saloc->nloc;
		//gradacc is used for accumulation in geom mode and for output in phy mode
		int ng=parms->powfs[ipowfs].ng;
		initzero(cuwfs[iwfs].gradacc, nsa*ng, 1);
		initzero(cuwfs[iwfs].gradcalc, nsa*ng, 1);
		if(parms->powfs[ipowfs].usephy||parms->powfs[ipowfs].dither){
			if(!cuwfs[iwfs].ints){
				if(parms->powfs[ipowfs].type==WFS_PY){//PYWFS
					cuwfs[iwfs].ints=curcell(1, 1, nsa, powfs[ipowfs].pywfs->cfg->nside);
				} else{
					cuwfs[iwfs].ints=curcell(nsa, 1, powfs[ipowfs].pixpsax, powfs[ipowfs].pixpsay);
				}
			} else{
				cuzero(cuwfs[iwfs].ints);
			}
		}
		if(parms->powfs[ipowfs].pistatout){
			if(parms->powfs[ipowfs].pistatstc){
				error("pistatstc is not supported yet.\n");
			}
			if(!cuwfs[iwfs].pistatout){
				const int notfx=powfs[ipowfs].notfx;/*necessary size to build detector image. */
				const int notfy=powfs[ipowfs].notfy;
				const int npsf=MAX(notfx, notfy);
				cuwfs[iwfs].pistatout=curcell(nsa, parms->powfs[ipowfs].nwvl, npsf, npsf);
			} else{
				cuzero(cuwfs[iwfs].pistatout);
			}
		}
		if(parms->powfs[ipowfs].i0save){
			cuzero(cuwfs[iwfs].intsout);
		}
		CUDA_SYNC_DEVICE;
	}
}
void gpu_wfssurf2gpu(const parms_t* parms, powfs_t* powfs){
	for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
		gpu_set(cuglobal->wfsgpu[iwfs]);
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		int ipowfs=parms->wfs[iwfs].powfs;
		int wfsind=parms->powfs[ipowfs].wfsind->p[iwfs];
		if(powfs[ipowfs].opdadd&&powfs[ipowfs].opdadd->p[wfsind]){
			cp2gpu(cuwfs[iwfs].opdadd, powfs[ipowfs].opdadd->p[wfsind]);
		}
	}
}
__global__ static void setup_rand(hiprandState* rstat, int seed){
	int id=threadIdx.x+blockIdx.x*blockDim.x;
	hiprand_init(seed, id, 0, &rstat[id]);
}
/**
   Seed the random number genrator
*/
void gpu_wfsgrad_seeding(const parms_t* parms, const powfs_t* powfs, rand_t* rstat){
	for(int iwfs=0; iwfs<parms->nwfs; iwfs++){
		gpu_set(cuglobal->wfsgpu[iwfs]);
		Array<cuwfs_t>& cuwfs=cuglobal->wfs;
		int seed=lrand(rstat);/*don't put this after continue. */
		int ipowfs=parms->wfs[iwfs].powfs;
		if(!parms->powfs[ipowfs].noisy) continue;
		int nsa=powfs[ipowfs].saloc->nloc*2;
		if(nsa<RAND_THREAD){
			cuwfs[iwfs].custatt=nsa;//number of threads
			cuwfs[iwfs].custatb=1;//number of blocks
		} else if(nsa<RAND_THREAD*RAND_BLOCK){
			cuwfs[iwfs].custatt=RAND_THREAD;
			cuwfs[iwfs].custatb=nsa/RAND_THREAD+(nsa%RAND_THREAD)?1:0;
		} else{
			cuwfs[iwfs].custatt=RAND_THREAD;
			cuwfs[iwfs].custatb=RAND_BLOCK;
		}
		DO(hipMalloc(&cuwfs[iwfs].custat, (cuwfs[iwfs].custatt*cuwfs[iwfs].custatb)*sizeof(hiprandState)));
		setup_rand<<<cuwfs[iwfs].custatb, cuwfs[iwfs].custatt>>>(cuwfs[iwfs].custat, seed);
	}
	CUDA_SYNC_DEVICE;
	gpu_print_mem("wfs seeding");
}
