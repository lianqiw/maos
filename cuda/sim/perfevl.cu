#include "hip/hip_runtime.h"
/*
  Copyright 2009-2025 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "../math/cumath.h"
#include "accphi.h"
#include "cudata.h"
#include "perf.h"
#if  defined(__cplusplus) && !USE_CPP
extern "C"{
#endif
#include "../../maos/utils.h"
#include "../../maos/ahst.h"
#if  defined(__cplusplus) && !USE_CPP
}
#endif
#undef TIMING
#define TIMING 0
#if !TIMING
#undef TIC
#undef tic
#undef toc
#define TIC
#define tic
#define ctoc(A)
#else
#define ctoc(A) toc2(A)
#endif
/**
	save aper_locs, aper_amp to GPU.
*/
const int TT_NBX=128;//Number of thread in a block. (for reduction).
__global__ static void calc_ptt_do(
	Real* cc,						/**<[out] coefficient*/
	const Real(*restrict loc)[2],	/**<[in] location of points*/
	const int nloc,					/**<[in] number of points*/
	const Real* restrict phi,		/**<[in] phase at every point*/
	const Real* restrict amp		/**<[in] amplitude at every point*/
	){
	__shared__ Real ccb[4][TT_NBX];
#if CUDA_VERSION<900 //supports GTX580 that often errors out in this routine
	repeat:
#endif
	for(int i=0; i<4; i++){
		ccb[i][threadIdx.x]=0.f;
	}
	int istep=blockDim.x*gridDim.x;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=istep){
		const Real tmp=phi[i]*amp[i];
		ccb[0][threadIdx.x]+=tmp*phi[i];
		ccb[1][threadIdx.x]+=tmp;
		ccb[2][threadIdx.x]+=tmp*loc[i][0];
		ccb[3][threadIdx.x]+=tmp*loc[i][1];
	}
	for(int step=(blockDim.x>>1);step>0;step>>=1){
		__syncthreads();
		if(threadIdx.x<step){
			for(int i=0; i<4; i++){
				ccb[i][threadIdx.x]+=ccb[i][threadIdx.x+step];
			}
		}
	}
#if CUDA_VERSION<900 //supports GTX580 that often errors out in this routine
//result sanity check
	__shared__ err;
	if(threadIdx.x==0) err=0;
	__syncthreads();
	if(threadIdx.x<4&&fabs(ccb[threadIdx.x][0])>1){
		err=1;
		printf("ccb[%d]=%g has wrong value, repeat\n", threadIdx.x, ccb[threadIdx.x][0]);
	}
	__syncthreads();
	if(err) goto repeat;
#endif
	if(threadIdx.x<4){
		atomicAdd(&cc[threadIdx.x], ccb[threadIdx.x][0]);
	}
}
//wraps calc_ptt_do
static void calc_ptt(Real* cc,
	const Real(*restrict loc)[2],
	const int nloc,
	const Real* restrict phi,
	const Real* restrict amp, stream_t& stream){
	DO(hipMemsetAsync(cc, 0, 4*sizeof(Real), stream));
	calc_ptt_do<<<DIM(nloc, TT_NBX), 0, stream>>>
		(cc, loc, nloc, phi, amp);
}
/*
  Let M be the modal matrix of pistion/tip/tilt. Calculate M'*diag(amp)*phi
  where amp is the amptliude weighting.  */
static int calc_ptt_post(real* rmsout, real* coeffout,
	const real ipcc, const dmat* imcc,
	const Real* ccb){
	real coeff[3];
	real tot=ccb[0];
	coeff[0]=ccb[1]; coeff[1]=ccb[2]; coeff[2]=ccb[3];
	if(coeffout){
		dmulvec3(coeffout, imcc, coeff);
	}
	int ans=0;
	if(rmsout){
		real pis=ipcc*coeff[0]*coeff[0];/*piston mode variance */
		real ptt=dwdot(coeff, imcc, coeff);/*p/t/t mode variance. */
		rmsout[0]=tot-pis;/*PR */
		rmsout[1]=ptt-pis;/*TT */
		rmsout[2]=tot-ptt;/*PTTR*/
		if(tot+1e-18<pis||tot+1e-18<ptt){//sanity check. allow round off error
			warning("tot=%g, pis=%g, ptt=%g\n", tot, pis, ptt);
			ans=1;
		}
	}
	return ans;
}

__global__ static void calc_ngsmod_do(Real* cc,
	const Real(*restrict loc)[2],
	const int nloc,
	const Real* restrict phi,
	const Real* restrict amp){
	__shared__ Real ccb[7][TT_NBX];
#if CUDA_VERSION<900 //supports GTX580 that often errors out in this routine
repeat:
#endif
#pragma unroll
	for(int i=0; i<7; i++){
		ccb[i][threadIdx.x]=0.f;
	}
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=blockDim.x*gridDim.x){
		const Real tmp=phi[i]*amp[i];
		const Real x=loc[i][0];
		const Real y=loc[i][1];
		ccb[0][threadIdx.x]+=tmp*phi[i];
		ccb[1][threadIdx.x]+=tmp;
		ccb[2][threadIdx.x]+=tmp*x;
		ccb[3][threadIdx.x]+=tmp*y;
		ccb[4][threadIdx.x]+=tmp*x*x;
		ccb[5][threadIdx.x]+=tmp*y*y;
		ccb[6][threadIdx.x]+=tmp*x*y;
	}
	for(int step=(blockDim.x>>1);step>0;step>>=1){
		__syncthreads();
		if(threadIdx.x<step){
#pragma unroll
			for(int i=0; i<7; i++){
				ccb[i][threadIdx.x]+=ccb[i][threadIdx.x+step];
			}
		}
	}
	__syncthreads();
#if CUDA_VERSION<900 //supports GTX580 that often errors out in this routine
	//result sanity check
	__shared__ err;
	if(threadIdx.x==0) err=0;
	__syncthreads();
	if(threadIdx.x<7 && fabs(ccb[threadIdx.x][0])>1){
		err=1;
		printf("ccb[%d]=%g has wrong value, repeat\n", threadIdx.x, ccb[threadIdx.x][0]);
	}
	__syncthreads();
	if(err) goto repeat;
#endif
	if(threadIdx.x<7){
		atomicAdd(&cc[threadIdx.x], ccb[threadIdx.x][0]);
	}
}
static void calc_ngsmod(Real* cc,
	const Real(*restrict loc)[2],
	const int nloc,
	const Real* restrict phi,
	const Real* restrict amp,
	stream_t& stream){
	DO(hipMemsetAsync(cc, 0, 7*sizeof(Real), stream));
	calc_ngsmod_do<<<DIM(nloc, TT_NBX), 0, stream>>>
		(cc, loc, nloc, phi, amp);
}


__global__ static void
strehlcomp_do(Comp* strehlc,
	const Real* opd, const Real* amp, const int nloc, const Real kk){
	extern __shared__ Real sbx[];
	Real* sby=sbx+blockDim.x;
	sbx[threadIdx.x]=0;
	sby[threadIdx.x]=0;
	Real s, c;
	for(int i=blockIdx.x*blockDim.x+threadIdx.x; i<nloc; i+=blockDim.x*gridDim.x){
		Z(sincos)(kk*opd[i], &s, &c);
		sbx[threadIdx.x]+=amp[i]*c;
		sby[threadIdx.x]+=amp[i]*s;
	}
	for(int step=(blockDim.x>>1);step>0;step>>=1){
		__syncthreads();
		if(threadIdx.x<step){
			sbx[threadIdx.x]+=sbx[threadIdx.x+step];
			sby[threadIdx.x]+=sby[threadIdx.x+step];
		}
	}
	if(threadIdx.x==0){
		if(strehlc){
			atomicAdd((Real*)strehlc, sbx[0]);
			atomicAdd((Real*)strehlc+1, sby[0]);
		}
		//donot try to accumuate x*x+y*y. that is not correct because of many blocks.
	}
}
/**
   Compute complex PSF and return.
*/
static void psfcomp(cuccell psfs, const curmat& iopdevl, int nwvl, int ievl, int nloc, hipStream_t stream){
	cucmat wvf;
	for(int iwvl=0; iwvl<nwvl; iwvl++){
		cucmat& psf=psfs[iwvl];
		if(!psf){
			psf=cucmat(cuglobal->perf.psfsize[iwvl], cuglobal->perf.psfsize[iwvl]);
		}
		if(cuglobal->perf.psfsize[iwvl]==1){
			strehlcomp_do<<<REDUCE(nloc), DIM_REDUCE*sizeof(Comp), stream>>>
				(psf(), iopdevl(), cudata->perf.amp, nloc, 2.*M_PI/cuglobal->perf.wvls[iwvl]);
		} else{
			if(wvf.Nx()!=cuglobal->perf.nembed[iwvl]){
				wvf=cucmat(cuglobal->perf.nembed[iwvl], cuglobal->perf.nembed[iwvl]);
			} else{
				cuzero(wvf, stream);
			}
			embed_wvf_do<<<DIM(iopdevl.Nx(), 256), 0, stream>>>
				(wvf(), iopdevl(), cudata->perf.amp, cudata->perf.embed[iwvl], nloc, cuglobal->perf.wvls[iwvl]);
			CUFFT(cuglobal->perf.plan[iwvl+nwvl*ievl], wvf(), HIPFFT_FORWARD);
			if(cuglobal->perf.psfsize[iwvl]<cuglobal->perf.nembed[iwvl]){
				corner2center_do<<<DIM2(psf.Nx(), psf.Ny(), 16), 0, stream>>>
					(psf(), psf.Nx(), psf.Ny(), wvf(), wvf.Nx(), wvf.Ny());
			} else{
				fftshift_do<<<DIM2(psf.Nx(), psf.Ny(), 16), 0, stream>>>
					(psf(), psf.Nx(), psf.Ny());
			}
		}
	}
}
/**
   Compute only PSF and add to result.
*/
static void psfcomp_r(curmat* psf, const curmat& iopdevl, int nwvl, int ievl, int nloc, int atomic, hipStream_t stream){
	cucmat wvf;
	for(int iwvl=0; iwvl<nwvl; iwvl++){
	//cucmat &wvf=cudata->perf.wvf[iwvl];
		if(wvf.Nx()!=cuglobal->perf.nembed[iwvl]){
			wvf=cucmat(cuglobal->perf.nembed[iwvl], cuglobal->perf.nembed[iwvl]);
		} else{
			cuzero(wvf, stream);
		}
		if(!psf[iwvl]) psf[iwvl]=curmat(cuglobal->perf.psfsize[iwvl], cuglobal->perf.psfsize[iwvl]);
		if(cuglobal->perf.psfsize[iwvl]==1){
			strehlcomp_do<<<REDUCE(nloc), DIM_REDUCE*sizeof(Real)*2, stream>>>
				(wvf(), iopdevl(), cudata->perf.amp, nloc, 2.*M_PI/cuglobal->perf.wvls[iwvl]);
				//do abs2.
			addcabs2_do<<<1, 1, 0, stream>>>(psf[iwvl](), 1.f, wvf(), 1.f, 1);
		} else{
			embed_wvf_do<<<DIM(iopdevl.Nx(), 256), 0, stream>>>
				(wvf(), iopdevl(), cudata->perf.amp, cudata->perf.embed[iwvl], nloc, cuglobal->perf.wvls[iwvl]);
			CUFFT(cuglobal->perf.plan[iwvl+nwvl*ievl], wvf(), HIPFFT_FORWARD);
			if(atomic){
				corner2center_abs2_atomic_do<<<DIM2((psf[iwvl]).Nx(), (psf[iwvl]).Ny(), 16), 0, stream>>>
					((psf[iwvl])(), (psf[iwvl]).Nx(), (psf[iwvl]).Ny(), wvf(), wvf.Nx(), wvf.Ny());
			} else{
				corner2center_abs2_do<<<DIM2((psf[iwvl]).Nx(), (psf[iwvl]).Ny(), 16), 0, stream>>>
					((psf[iwvl])(), (psf[iwvl]).Nx(), (psf[iwvl]).Ny(), wvf(), wvf.Nx(), wvf.Ny());
			}
		}
	}
}
#define PERFEVL_WFE_GPU(cc,ccb)						\
    if((parms->evl.split && recon->ngsmod->nmod==2)			\
       || (!parms->evl.split && parms->evl.nmod==3)){			\
	calc_ptt(cc, cudata->perf.locs(), nloc, iopdevl(), cudata->perf.amp, stream); \
	DO(hipMemcpyAsync(ccb, cc, 4*sizeof(Real), D2H, stream)); \
    }else if(parms->evl.split){					\
	calc_ngsmod(cc, cudata->perf.locs(), nloc, iopdevl(), cudata->perf.amp, stream); \
	DO(hipMemcpyAsync(ccb, cc, 7*sizeof(Real), D2H, stream)); \
    }

#define PERFEVL_WFE_CPU(ans, pclep, pclmp, cleNGSmp, ccb)		\
    if(nmod!=3){									\
	TO_IMPLEMENT;/*mode decomposition. */			\
    }												\
    int ans=0;										\
    if(parms->evl.split){							\
	real *pcleNGSmp=PCOL(cleNGSmp->p[ievl], isim);	\
	real coeff[6];/*convert precision*/				\
	coeff[0]=ccb[1]; coeff[1]=ccb[2];				\
	coeff[2]=ccb[3]; coeff[3]=ccb[4];				\
	coeff[4]=ccb[5]; coeff[5]=ccb[6];				\
	ans=ngsmod_dot_post(nmod==3?pclep:0, nmod==3?pclmp:0,		\
			 pcleNGSmp,ccb[0],coeff,recon->ngsmod, aper,thetax,thetay); \
    }else{								\
	ans=calc_ptt_post(pclep, pclmp, aper->ipcc, aper->imcc, ccb);	\
    }
static void save_cov_opdmean(curmat& opdcov, curmat& opdmean, curmat& iopdevl,
							dmat **opdcov_cpu, dmat **opdmean_cpu,
							int ingpu, int do_cov, int do_opdmean, stream_t& stream){
	if(ingpu){
		if(do_cov){
			cugemm(opdcov, (Real)1, iopdevl, iopdevl, "nt", (Real)1, stream);
		}
		if(do_opdmean){
			Add(opdmean, (Real)1., iopdevl, (Real)1., stream);
		}
	} else{//accumulate in CPU to save GPU memory.
		dmat* tmp=NULL;
		cp2cpu(&tmp, iopdevl, stream);
		if(do_cov){
			dmm(opdcov_cpu, (Real)1, tmp, tmp, "nt", 1);
		}
		if(do_opdmean){
			dadd(opdmean_cpu, (Real)1, tmp, (Real)1);
		}
		dfree(tmp);
	}

}
/**
   Performance evaluation. Designed to replace perfevl_ievl in maos/perfevl.c
*/
void* gpu_perfevl_queue(thread_t* info){
	TIC;tic;
	sim_t* simu=(sim_t*)info->data;
	const parms_t* parms=simu->parms;
	const aper_t* aper=simu->aper;
	const recon_t* recon=simu->recon;
	const int isim=simu->perfisim;
	const int imoao=parms->evl.moao;
	const int nloc=aper->locs->nloc;
	const int nwvl=parms->evl.nwvl;
	for(int ievl=info->start; ievl<info->end; ievl++){
		gpu_set(cuglobal->evlgpu[ievl]);
		//info("thread %ld gpu %d ievl %d start\n", thread_id(), cudata->igpu, ievl);
		const int do_psf_cov=(parms->evl.psfmean||parms->evl.psfhist||parms->evl.cov||parms->evl.opdmean)
			&&isim>=parms->evl.psfisim&&parms->evl.psf->p[ievl]!=0;
		const int save_evlopd=parms->save.evlopd>0&&((isim+1)%parms->save.evlopd)==0;
		const real thetax=parms->evl.thetax->p[ievl];
		const real thetay=parms->evl.thetay->p[ievl];

		stream_t& stream=cudata->perf_stream;
		curmat& iopdevl=cuglobal->perf.opd[ievl];
		// iopdevl must be in device memory. 6 times slower if in host memory.
		if(cuglobal->perf.surf&&cuglobal->perf.surf[ievl]){
			Copy(iopdevl, cuglobal->perf.surf[ievl], stream);
		} else{
			iopdevl.Zero(stream);
		}
		if(parms->sim.idealevl){
			mapcell2loc(iopdevl(), cudata->perf.locs_dm[ievl], cudata->dmproj, 
				parms->evl.hs->p[ievl], 0, thetax, thetay, 0, 0, 1, stream);
		} else if(simu->atm&&!parms->sim.wfsalias){
			atm2loc(iopdevl(), cudata->perf.locs, parms->evl.hs->p[ievl], 0, thetax, thetay,
				0, 0, parms->sim.dt, isim, 1, stream);
		}
		if(simu->telws){//Wind shake
			Real tt=simu->telws->p[isim];
			Real angle=simu->winddir?simu->winddir->p[0]:0;
			curaddptt(iopdevl, cudata->perf.locs(), 0, tt*cosf(angle), tt*sinf(angle), stream);
		}
		if(simu->telfocusreal){
			Real focus=-simu->telfocusreal->p[0]->p[0];
			add_focus_do<<<DIM(nloc, 256), 0, stream>>>(iopdevl, cudata->perf.locs(), nloc, focus);
		}
		if(save_evlopd && simu->save->evlopdol){
			zfarr_push_scale(simu->save->evlopdol[ievl], isim, iopdevl, 1, stream);
		}
		if(parms->plot.run&&isim%parms->plot.run==0){
			drawopdamp_gpu("Evlol", aper->locs, iopdevl, stream, aper->amp1, 0,
				"Science Open Loop OPD", "x (m)", "y (m)", "OL %d", ievl);
		}
		PERFEVL_WFE_GPU(cuglobal->perf.cc_ol[ievl](), cuglobal->perf.ccb_ol[ievl]);
		if((parms->evl.psfmean||parms->evl.cov||parms->evl.opdmean)
			&&isim>=parms->evl.psfisim
			&&((parms->evl.psfol==1&&ievl==parms->evl.indoa)
				||(parms->evl.psfol==2&&parms->evl.psf->p[ievl]))){
			  //calculate Openloop PSF. we also test psfisim to synchronize with psfcl.
			curmat opdcopy;
			curmv(cuglobal->perf.coeff[ievl](), 0, cudata->perf.imcc,
				cuglobal->perf.cc_ol[ievl](), 'n', 1, stream);
			Copy(opdcopy, iopdevl, stream);
			if(parms->evl.pttr->p[ievl]){//remove piston/tip/tilt
				curaddptt(opdcopy, cudata->perf.locs(), cuglobal->perf.coeff[ievl](), -1, -1, -1, stream);
				warning_once("Removing piston/tip/tilt from OPD.\n");
			} else{//remove piston only
				curaddptt(opdcopy, cudata->perf.locs(), cuglobal->perf.coeff[ievl](), -1, 0, 0, stream);
			}
			if(parms->evl.cov||parms->evl.opdmean){
				save_cov_opdmean(cudata->perf.opdcovol, cudata->perf.opdmeanol, opdcopy,
								&simu->evlopdcovol, &simu->evlopdmeanol,
								parms->gpu.psf, parms->evl.cov, parms->evl.opdmean, stream);
			}
			if(parms->evl.psfmean){
				psfcomp_r(cudata->perf.psfol(), opdcopy, nwvl, ievl, nloc, parms->evl.psfol==2?1:0, stream);
				if(parms->plot.run&&isim%parms->plot.run==0){
					int count=parms->gpu.psf?(simu->perfisim+1-parms->evl.psfisim):1;
					if(parms->evl.psfol==2){
						count*=lsum(parms->evl.psf);
					}

					for(int iwvl=0; iwvl<nwvl; iwvl++){
						drawpsf_gpu("PSFol", cudata->perf.psfol[iwvl], count, stream,
							parms->plot.psf==1?1:0, parms->plot.psfmin, "Science Open Loop PSF",
							"x", "y", "OL%2d %.2f", ievl, parms->evl.wvl->p[iwvl]*1e6);
					}
				}
				if(!parms->gpu.psf){ //need to move psf from GPU to CPU for accumulation.
					for(int iwvl=0; iwvl<nwvl; iwvl++){
						add2cpu(&simu->evlpsfolmean->p[iwvl], 1, cudata->perf.psfol[iwvl], 1, stream);
						cuzero(cudata->perf.psfol[iwvl]); //do not accumulate in gpu.
					} 
				}
			}
		}
		if(parms->sim.evlol) continue;
		if(parms->evl.tomo){
			if(parms->tomo.square){
				mapcell2loc(iopdevl(), cudata->perf.locs, cudata->opdr, 
					parms->evl.hs->p[ievl], 0, thetax, thetay,
					0, 0, -1, stream);
			}else{
				TO_IMPLEMENT;
			}
		} else{
			wait_dmreal(simu, simu->perfisim);
			mapcell2loc(iopdevl(), cudata->perf.locs_dm[ievl], cudata->dmreal, 
				parms->evl.hs->p[ievl], 0, thetax, thetay,
				0, 0, -1, stream);
			if(simu->ttmreal){
				curaddptt(iopdevl, cudata->perf.locs(), 0, -simu->ttmreal->p[0], -simu->ttmreal->p[1], stream);
			}
			if(imoao!=-1){
				mapcell2loc(iopdevl(), cudata->perf.locs, cudata->dm_evl[ievl], 
					INFINITY, 0, 0, 0, 0, 0, -1, stream);
			}
		}
		if(save_evlopd){
			zfarr_push_scale(simu->save->evlopdcl[ievl], isim, iopdevl, 1, stream);
		}

		if(parms->plot.run&&isim%parms->plot.run==0){
			drawopdamp_gpu("Evlcl", aper->locs, iopdevl, stream, aper->amp1, 0,
				"Science Closed loop OPD", "x (m)", "y (m)", "CL %d", ievl);
		}
		PERFEVL_WFE_GPU(cuglobal->perf.cc_cl[ievl](), cuglobal->perf.ccb_cl[ievl]);
		if(do_psf_cov&&(parms->evl.psf->p[ievl]&1)){//also do normal psf
			curmv(cuglobal->perf.coeff[ievl](), 0, cudata->perf.imcc,
				cuglobal->perf.cc_cl[ievl](), 'n', 1, stream);
			if(parms->evl.pttr->p[ievl]){
				curaddptt(iopdevl, cudata->perf.locs(), cuglobal->perf.coeff[ievl], -1, -1, -1, stream);
			} else{
				curaddptt(iopdevl, cudata->perf.locs(), cuglobal->perf.coeff[ievl], -1, 0, 0, stream);
			}
			if(parms->evl.cov || parms->evl.opdmean){
				save_cov_opdmean(cuglobal->perf.opdcov[ievl], cuglobal->perf.opdmean[ievl], iopdevl,
								&simu->evlopdcov->p[ievl], &simu->evlopdmean->p[ievl],
								parms->gpu.psf, parms->evl.cov, parms->evl.opdmean, stream);
			}//opdcov
			if(parms->evl.psfhist||parms->evl.psfmean){
				if(parms->evl.psfhist){
					//Compute complex.
					cuccell psfs(nwvl, 1);
					psfcomp(psfs, iopdevl, nwvl, ievl, nloc, stream);
					zfarr_push_scale(simu->save->evlpsfhist[ievl], isim, psfs, 1, stream);
					if(parms->evl.psfmean){
						for(int iwvl=0; iwvl<nwvl; iwvl++){
							curaddcabs2(cuglobal->perf.psfcl[iwvl+nwvl*ievl], 1,
								psfs[iwvl], 1, stream);
						}
					}
				} else if(parms->evl.psfmean){
					psfcomp_r(cuglobal->perf.psfcl+nwvl*ievl, iopdevl, nwvl, ievl, nloc, 0, stream);
				}
				if(parms->plot.run&&isim%parms->plot.run==0){
					int count=parms->gpu.psf?(simu->perfisim+1-parms->evl.psfisim):1;
					for(int iwvl=0; iwvl<nwvl; iwvl++){
						drawpsf_gpu("PSFcl", cuglobal->perf.psfcl[iwvl+nwvl*ievl], count, stream,
							parms->plot.psf==1?1:0, parms->plot.psfmin, "Science Closed Loop PSF",
							"x", "y", "CL%2d %.2f", ievl, parms->evl.wvl->p[iwvl]*1e6);
					}
				}
				if(!parms->gpu.psf){
					for(int iwvl=0; iwvl<nwvl; iwvl++){
						add2cpu(&simu->evlpsfmean->p[iwvl+ievl*nwvl], 1, cuglobal->perf.psfcl[iwvl+ievl*nwvl], 1, stream);
						cuzero(cuglobal->perf.psfcl[iwvl+ievl*nwvl]);
					}
				}
			}
		}
		//info("thread %ld gpu %d ievl %d queued\n", thread_id(), cudata->igpu, ievl);
		ctoc("queued");
	}//for ievl
	return NULL;
}
void* gpu_perfevl_sync(thread_t* info){
	TIC;tic;
	sim_t* simu=(sim_t*)info->data;
	const parms_t* parms=simu->parms;
	const int isim=simu->perfisim;
	const aper_t* aper=simu->aper;
	const recon_t* recon=simu->recon;
	const int nmod=parms->evl.nmod;
	for(int ievl=info->start; ievl<info->end; ievl++){
		gpu_set(cuglobal->evlgpu[ievl]);
		hipStream_t stream=cudata->perf_stream;
		const real thetax=parms->evl.thetax->p[ievl];
		const real thetay=parms->evl.thetay->p[ievl];
		/*Setup pointers for easy usage */
		real* polmp=PCOL(simu->olmp->p[ievl], isim);
		real* pclmp=PCOL(simu->clmp->p[ievl], isim);
		real* polep=PCOL(simu->olep->p[ievl], isim);
		real* pclep=PCOL(simu->clep->p[ievl], isim);
		CUDA_SYNC_STREAM;
		post_dmreal(simu);
		PERFEVL_WFE_CPU(ans1, polep, polmp, simu->oleNGSmp, cuglobal->perf.ccb_ol[ievl]);
		PERFEVL_WFE_CPU(ans2, pclep, pclmp, simu->cleNGSmp, cuglobal->perf.ccb_cl[ievl]);
		if(ans1||ans2){
			static int status=0;
			if(status>2){
				error("Step %5d: Perfevl fails, quite after %d redos\n", isim, status);
			}else{
				warning("Step %5d: Perfevl fails, redo\n", isim);
			}
			status++;
			gpu_perfevl_queue(info);
			gpu_perfevl_sync(info);
		}
		//info("thread %ld gpu %d ievl %d end\n", thread_id(), cudata->igpu, ievl);
	}//for ievl
	ctoc("gpu_perfevl_sync");
	return NULL;
}

/**
   Compute the PSF or OPDCOV for NGS mode removed opd.
*/
void gpu_perfevl_ngsr(sim_t* simu, real* cleNGSm){
	const parms_t* parms=simu->parms;
	const aper_t* aper=simu->aper;
	const int nloc=aper->locs->nloc;
	const int nwvl=parms->evl.nwvl;
OMP_TASK_FOR(4)	
	for(int ievl=0; ievl<parms->evl.nevl; ievl++){
		if((parms->evl.psf->p[ievl] &2)){
			//warning_once("Compare with CPU code to verify accuracy. Need to verify focus mode\n");
			gpu_set(cuglobal->evlgpu[ievl]);
			curmat& iopdevl=cuglobal->perf.opd[ievl];
			stream_t& stream=cudata->perf_stream;
			ngsmod2loc(iopdevl, cudata->perf.locs(), simu->recon->ngsmod, cleNGSm,
				parms->evl.thetax->p[ievl], parms->evl.thetay->p[ievl],
				-1, stream);
			if(parms->plot.run&&simu->perfisim%parms->plot.run==0){
				drawopdamp_gpu("Evlcl", aper->locs, iopdevl, stream, aper->amp1, 0,
					"Science Closed loop OPD", "x (m)", "y (m)", "ngsr %d", ievl);
			}
			if(parms->evl.pttr->p[ievl]){
				calc_ptt(cuglobal->perf.cc_cl[ievl](), cudata->perf.locs(), nloc, iopdevl(), cudata->perf.amp, stream);
				DO(hipMemcpyAsync(cuglobal->perf.ccb_cl[ievl], cuglobal->perf.cc_cl[ievl](),
					4*sizeof(Real), D2H, stream));
				CUDA_SYNC_STREAM;
				real ptt[3]={0,0,0};
				calc_ptt_post(NULL, ptt, aper->ipcc, aper->imcc, cuglobal->perf.ccb_cl[ievl]);
				curaddptt(iopdevl, cudata->perf.locs(), -ptt[0], -ptt[1], -ptt[2], stream);
			}
			if(parms->evl.cov || parms->evl.opdmean){
				save_cov_opdmean(cuglobal->perf.opdcov_ngsr[ievl], cuglobal->perf.opdmean_ngsr[ievl], iopdevl,
								&simu->evlopdcov_ngsr->p[ievl], &simu->evlopdmean_ngsr->p[ievl],
								parms->gpu.psf, parms->evl.cov, parms->evl.opdmean, stream);
			}/*opdcov */
			if(parms->evl.psfhist||parms->evl.psfmean){
				if(parms->evl.psfhist){
					/*Compute complex. */
					cuccell psfs(nwvl, 1);
					psfcomp(psfs, iopdevl, nwvl, ievl, nloc, stream);
					zfarr_push_scale(simu->save->evlpsfhist_ngsr[ievl], simu->perfisim, psfs, 1, stream);
					if(parms->evl.psfmean){
						for(int iwvl=0; iwvl<nwvl; iwvl++){
							curaddcabs2(cuglobal->perf.psfcl_ngsr[iwvl+nwvl*ievl], 1,
								psfs[iwvl], 1, stream);
						}
					}
				} else if(parms->evl.psfmean){
					psfcomp_r(cuglobal->perf.psfcl_ngsr+nwvl*ievl, iopdevl, nwvl, ievl, nloc, 0, stream);
				}
				if(parms->plot.run&&simu->perfisim%parms->plot.run==0){
					int count=parms->gpu.psf?(simu->perfisim+1-parms->evl.psfisim):1;
					for(int iwvl=0; iwvl<nwvl; iwvl++){
						drawpsf_gpu("PSFngsr", cuglobal->perf.psfcl_ngsr[iwvl+nwvl*ievl], count, stream,
							parms->plot.psf==1?1:0, parms->plot.psfmin, "Science Closed Loop PSF",
							"x", "y", "CL%2d %.2f", ievl, parms->evl.wvl->p[iwvl]*1e6);
					}
				}
				if(!parms->gpu.psf){
					for(int iwvl=0; iwvl<nwvl; iwvl++){
						add2cpu(&simu->evlpsfmean_ngsr->p[iwvl+ievl*nwvl], 1, cuglobal->perf.psfcl_ngsr[iwvl+ievl*nwvl], 1, stream);
						cuzero(cuglobal->perf.psfcl_ngsr[iwvl+ievl*nwvl]);
					}
				}
			}
			CUDA_SYNC_STREAM;
		}
	}
}
void gpu_perfevl_save(sim_t* simu){
	const parms_t* parms=simu->parms;
	if(!parms->evl.nevl) return;
	const int isim=simu->perfisim;
	if(parms->evl.psfmean&&CHECK_SAVE(parms->evl.psfisim, parms->sim.end, isim, parms->evl.psfmean)){
		info("Step %d: Output PSF\n", isim);
		const int nwvl=parms->evl.nwvl;
		int nacc=(simu->perfisim+1-parms->evl.psfisim);//total accumulated.
		const real scale=1./(real)nacc;
		if(cudata->perf.psfol){
			const real scaleol=(parms->evl.psfol==2)?(scale/parms->evl.npsf):(scale);
			/*copy the PSF accumulated in all the GPUs to CPU.*/
			X(cell)* temp=X(cellnew)(nwvl, 1);
			X(cell)* temp2=X(cellnew)(nwvl, 1);
			for(int im=0; im<NGPU; im++){
				gpu_set(im);
				cp2cpu(&temp2, cudata->perf.psfol, 0);
				hipStreamSynchronize(0);
				X(celladd)(&temp, 1, temp2, scaleol);
			}
			for(int iwvl=0; iwvl<nwvl; iwvl++){
				if(!temp||!temp->p[iwvl]) continue;
				temp->p[iwvl]->keywords=evl_keywords(simu->parms, simu->aper, -1, iwvl, isim);
				zfarr_push(simu->save->evlpsfolmean, isim*nwvl+iwvl, temp->p[iwvl]);
				//free(temp->p[iwvl]->keywords); temp->p[iwvl]->keywords=NULL;
			}
			X(cellfree)(temp);
			X(cellfree)(temp2);
		}
		if(cuglobal->perf.psfcl){
			for(int ievl=0; ievl<parms->evl.nevl; ievl++){
				if(!(parms->evl.psf->p[ievl]&1)) continue;
				gpu_set(cuglobal->evlgpu[ievl]);
				hipStream_t stream=cudata->perf_stream;
				for(int iwvl=0; iwvl<nwvl; iwvl++){
					curmat& pp=cuglobal->perf.psfcl[iwvl+nwvl*ievl];
					if(1){//update keywords for the exposure time
						char *tmp=evl_keywords(simu->parms, simu->aper, ievl, iwvl, isim);
						pp.keywords=tmp;
						free(tmp);
					}
					zfarr_push_scale(simu->save->evlpsfmean[ievl], isim*nwvl+iwvl, pp, scale, stream);
				}
			}
		}
		if(cuglobal->perf.psfcl_ngsr){
			for(int ievl=0; ievl<parms->evl.nevl; ievl++){
				if(!(parms->evl.psf->p[ievl]&2)) continue;
				gpu_set(cuglobal->evlgpu[ievl]);
				hipStream_t stream=cudata->perf_stream;
				for(int iwvl=0; iwvl<nwvl; iwvl++){
					curmat& pp=cuglobal->perf.psfcl_ngsr[iwvl+nwvl*ievl];
					if(1){//update keywords for the exposure time
						char *tmp=evl_keywords(simu->parms, simu->aper, ievl, iwvl, isim);
						pp.keywords=tmp;
						free(tmp);
					}
					zfarr_push_scale(simu->save->evlpsfmean_ngsr[ievl], isim*nwvl+iwvl, pp, scale, stream);
				}
			}
		}
	}
	//notice that evl.cov always enables evl.opdmean
	if(parms->evl.opdmean&&CHECK_SAVE(parms->evl.psfisim, parms->sim.end, isim, parms->evl.opdmean)){
		info("Step %d: Output opdcov\n", isim);
		int nacc=(simu->perfisim+1-parms->evl.psfisim);//total accumulated.
		const real scale=1./(real)nacc;
		for(int ievl=0; ievl<parms->evl.nevl; ievl++){
			if(!parms->evl.psf->p[ievl]) continue;
			gpu_set(cuglobal->evlgpu[ievl]);
			hipStream_t stream=cudata->perf_stream;
			if((parms->evl.psf->p[ievl]&1)){
				if(parms->evl.cov){
					curmat& pp=cuglobal->perf.opdcov[ievl];
					zfarr_push_scale(simu->save->evlopdcov[ievl], isim, pp, scale, stream);
				}
				{
					curmat& pp=cuglobal->perf.opdmean[ievl];
					zfarr_push_scale(simu->save->evlopdmean[ievl], isim, pp, scale, stream);
				}
			}
			if((parms->evl.psf->p[ievl]&2)){
				if(parms->evl.cov){
					curmat& pp=cuglobal->perf.opdcov_ngsr[ievl];
					zfarr_push_scale(simu->save->evlopdcov_ngsr[ievl], isim, pp, scale, stream);
				}
				{
					curmat& pp=cuglobal->perf.opdmean_ngsr[ievl];
					zfarr_push_scale(simu->save->evlopdmean_ngsr[ievl], isim, pp, scale, stream);
				}
			}
		}
		if(parms->evl.psfol){
			const real scaleol=(parms->evl.psfol==2)?(scale/parms->evl.npsf):(scale);
			if(parms->evl.cov){
				X(mat)* temp=NULL;
				X(mat)* temp2=NULL;
				for(int im=0; im<NGPU; im++){
					gpu_set(im);
					cp2cpu(&temp2, cudata->perf.opdcovol, 0);
					hipStreamSynchronize(0);
					X(add)(&temp, 1, temp2, scaleol);
				}
				zfarr_push(simu->save->evlopdcovol, isim, temp);
				X(free)(temp);
				X(free)(temp2);
			}
			{
				X(mat)* temp=NULL;
				X(mat)* temp2=NULL;
				for(int im=0; im<NGPU; im++){
					gpu_set(im);
					cp2cpu(&temp2, cudata->perf.opdmeanol, 0);
					hipStreamSynchronize(0);
					X(add)(&temp, 1, temp2, scaleol);
				}
				zfarr_push(simu->save->evlopdmeanol, isim, temp);
				X(free)(temp);
				X(free)(temp2);
			}
		}
	}
}
